//:: cases BasicCuda
//:: tool silicon
//:: verdict Pass

#include <hip/hip_runtime.h>

/*@
    context \pointer_index(a, threadIdx.x, write);
@*/
__global__ void example(int a[], int len) {
    int tid = threadIdx.x;
    a[tid] = tid;
    /*@
        context \pointer_index(a, threadIdx.x, write);
    @*/
    __syncthreads();
    a[tid] = a[tid] * 2;
}