#include "hip/hip_runtime.h"
//:: case ASSSPv3
//:: tool silicon
/***********************************************************************************
Created by Mohsen Safari.
************************************************************************************/
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

/*@
  yields seq<int> contrib; 
  given seq<int> cost_seq;
  given seq<int> oldcost_seq;
  given seq<int> start_seq;
  given seq<int> end_seq;
  given seq<int> weight_seq; 

  context_everywhere V == 1024 && A == 10 * V;
  context_everywhere opencl_gsize == V;
  context_everywhere 10 == opencl_gcount;
  context_everywhere source >= 0 && source < V;
  context_everywhere counter >= 0 && counter < V-1;

  kernel_invariant \pointer(g_start, A, 1\A);
  kernel_invariant \pointer(g_end, A, 1\A);
  kernel_invariant \pointer(g_weight, A, 1\A);
  kernel_invariant \pointer(g_cost, A, 1);

  requires \pointer_index(g_start, \gtid, 1\A);
  requires \pointer_index(g_end, \gtid, 1\A);
  requires \pointer_index(g_weight, \gtid, 1\A);
  requires \pointer_index(g_cost, \gtid, write);
@*/
__global__ void CUDAKernel(int* g_start, int* g_end, int* g_weight, int* g_cost, int V, int A, int counter, int source)
{
  int tid = blockIdx.x * V + threadIdx.x;
  //@ assert tid == \gtid;
  //@ assert 0 <= tid && tid < A;
  atomicRelax(g_cost+g_end[tid], g_weight[tid], g_cost[g_start[tid]]);
}
