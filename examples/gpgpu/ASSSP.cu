#include "hip/hip_runtime.h"
//:: case ASSSP

/* Currently this file does not pass because of some renaming errors. Mohsen Safari might have a version in which
 * data-race freedom is verified, and which was also used in his ASSSP paper. This file might be included in the future,
 * and can then replace this file.
 */

/***********************************************************************************
Created by Mohsen Safari.
************************************************************************************/
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

//@ pure int inf() = -1; //100000000; any negative integer

/*@ 
pure bool Graph(int V, int A, seq<int> start, seq<int> end, seq<int> weight) =
	0 < V &&
	0 < A && |start| == A && |end| == A && |weight| == A &&
	(\forall int i; 0 <= i && i < A; 
		0 <= start[i] && start[i] < V && 
		0 <= end[i] && end[i] < V && 
		start[i] != end[i] && 
		(\forall int j; 0 <= j && j < A && i != j && start[i] == start[j]; end[i] != end[j]) &&
     weight[i] > 0); 
@*/

/*@ 
requires Graph(V, A, start, end, weight);
ensures (\forall int i; i >= 0 && i < A; Path(V, A, start, end, weight, start[i], end[i], seq<int> { i }));
pure bool Path(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x, int y, seq<int> P) =
	0 <= x && x < V &&
	0 <= y && y < V &&
	(\forall int i; 0 <= i && i < |P|; 0 <= P[i] && P[i] < A) &&
	(0 == |P| ==> x == y) &&
	(0 < |P| ==> start[P[0]] == x && end[P[|P| - 1]] == y) &&
	(\forall int i; 0 <= i && i < |P| - 1; end[P[i]] == start[P[i + 1]]);
@*/

/*@ 
requires Graph(V, A, start, end, weight);
ensures (\forall int i; i >= 0 && i < A; SimplePath(V, A, start, end, weight, start[i], end[i], seq<int> { i }));
pure bool SimplePath(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x, int y, seq<int> P) =
  Path(V, A, start, end, weight, x, y, P) &&
  (\forall int i; 0 <= i && i < |P| - 1; (\forall int j; i < j && j < |P|; start[P[i]] != end[P[j]])); 
@*/

/*@ 
requires Graph(V, A, start, end, weight);
pure bool ExPath(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x, int y, int len) =
	(\exists seq<int> P; Path(V, A, start, end, weight, x, y, P); |P| <= len);
@*/

/*@   
requires Graph(V, A, start, end, weight);
ensures (\forall int i; i >= 0 && i < A; ExPathEqual(V, A, start, end, weight, start[i], end[i], weight[i]));
ensures ExPathEqual(V, A, start, end, weight, x, x, 0);
pure bool ExPathEqual(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x, int y, int len) =
	(\exists seq<int> P; Path(V, A, start, end, weight, x, y, P); cost(V, A, start, end, weight, P) == len);
@*/

/*@
requires Graph(V, A, start, end, weight);
requires (\forall int i; 0 <= i && i < |P|; 0 <= P[i] && P[i] < A);
ensures 0 <= \result;
ensures 0 == |P| ==> \result == 0;
ensures (P != seq<int>{}) ==> \result > 0;
ensures (\result == 0) ==> (P == seq<int>{});
ensures (\result != 0) ==> (P != seq<int>{});
ensures 1 == |P| ==> \result == weight[P[0]];
pure int cost(int V, int A, seq<int> start, seq<int> end, seq<int> weight, seq<int> P) =  
  0 < |P| ? weight[P[0]] + cost(V, A, start, end, weight, tail(P)) : 0;
@*/  

/*@
ensures (xs + ys) + zs == xs + (ys + zs);
pure bool iseq_assoc(seq<int> xs, seq<int> ys, seq<int> zs) = true;
@*/

/*@
requires Graph(V, A, start, end, weight);
requires Path(V, A, start, end, weight, x, y, P);
requires 0 <= a && a < A;
requires end[a] == x;
ensures \result && Path(V, A, start, end, weight, start[a], y, seq<int> { a } + P);
pure bool lemma_path_append(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x, int y, seq<int> P, int a);
@*/

/*@
requires Graph(V, A, start, end, weight);
requires Path(V, A, start, end, weight, s, t, P);
requires Path(V, A, start, end, weight, t, u, Q);
ensures \result && Path(V, A, start, end, weight, s, u, P + Q);
pure bool lemma_path_trans(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int s, int t, int u, seq<int> P, seq<int> Q) =
	0 < |P| ==> (
		lemma_path_trans(V, A, start, end, weight, end[P[0]], t, u, tail(P), Q) &&
		lemma_path_append(V, A, start, end, weight, end[P[0]], u, tail(P) + Q, P[0]) &&
		iseq_assoc(seq<int> { P[0] }, tail(P), Q) &&
		seq<int> { P[0] } + tail(P) == P
	); 
@*/

/*@
requires 0 < |xs|;
ensures (xs + ys)[0] == xs[0];
ensures tail(xs + ys) == tail(xs) + ys;
ensures \result;  
pure bool lemma_app_nonempty(seq<int> xs, seq<int> ys) = true; 
@*/

/*@
requires 0 < |ys|;
ensures \result && ((xs + seq<int>{ys[0]}) + tail(ys)) == xs + ys;
pure bool lemma_app_append_right(seq<int> xs, seq<int> ys) =
  iseq_assoc(xs, seq<int>{ys[0]}, tail(ys));
@*/

/*@
requires Graph(V, A, start, end, weight);
requires (\forall int i; 0 <= i && i < |P|; 0 <= P[i] && P[i] < A);
requires (\forall int i; 0 <= i && i < |Q|; 0 <= Q[i] && Q[i] < A);
ensures \result && cost(V, A, start, end, weight, P + Q) == cost(V, A, start, end, weight, P) + cost(V, A, start, end, weight, Q);
pure bool lemma_cost_app(int V, int A, seq<int> start, seq<int> end, seq<int> weight, seq<int> P, seq<int> Q) =  
  0 < |P| ==> lemma_app_nonempty(P, Q) && lemma_cost_app(V, A, start, end, weight, tail(P), Q);
@*/

/*@
requires Graph(V, A, start, end, weight);
requires Path(V, A, start, end, weight, s, t, P);
requires Path(V, A, start, end, weight, t, u, Q);
ensures \result && Path(V, A, start, end, weight, s, u, P + Q);
ensures \result && cost(V, A, start, end, weight, P + Q) == cost(V, A, start, end, weight, P) + cost(V, A, start, end, weight, Q);
pure bool lemma_cost_path(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int s, int t, int u, seq<int> P, seq<int> Q) =
  lemma_path_trans(V, A, start, end, weight, s, t, u, P, Q) &&
  lemma_cost_app(V, A, start, end, weight, P, Q);
@*/

/*@
requires b;
ensures \result == xs;
pure seq<int> assertSeq(seq<int> xs, bool b) = xs;
@*/

/*@
requires Graph(V, A, start, end, weight);
ensures \result && (\forall int i; i >= 0 && i < A; Path(V, A, start, end, weight, start[i], end[i], seq<int>{i}));
ensures	\result && (\forall int i; i >= 0 && i < A; ExPathEqual(V, A, start, end, weight, start[i], end[i], weight[i]));	
pure bool lemma_one_path(int V, int A, seq<int> start, seq<int> end, seq<int> weight) = true;
@*/


/* ** Invariant 10 */
/*@
requires Graph(V, A, start, end, weight);
requires ExPathEqual(V, A, start, end, weight, s, t, len1); 
requires ExPathEqual(V, A, start, end, weight, t, u, len2); 
ensures \result && ExPathEqual(V, A, start, end, weight, s, u, len1+len2);
pure bool lemma_expath_trans(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int s, int t, int u, int len1, int len2) =
	(\forall seq<int> P; Path(V, A, start, end, weight, s, t, P);
		(\forall seq<int> Q; Path(V, A, start, end, weight, t, u, Q);
			lemma_cost_path(V, A, start, end, weight, s, t, u, P, Q)));
@*/


/*@
  requires i >= 0 && i <= |xs|-1;
  ensures |\result| == |xs| - 1 - i;
  ensures (\forall int k; k >= 0 && k < |\result|; \result[k] == xs[i+k]);
  ensures i == 0 ==> (\forall int k; k >= 0 && k < |xs|-1; \result[k] == xs[k]);
  pure seq<int> subseq(seq<int> xs, int i) = (i != |xs|-1) ? seq<int>{xs[i]} + subseq(xs, i+1) : seq<int>{}; 
@*/

/* ** Invariant 12 */
/*@
requires Graph(V, A, start, end, weight);
requires |oldcost| == V && |cost| == V;
requires (\forall int k; 0 <= k && k < V; cost[k] == inf() ==> oldcost[k] == cost[k]);
requires (\forall int k; 0 <= k && k < V; oldcost[k] == inf() ==> oldcost[k] <= cost[k]);
requires (\forall int k; 0 <= k && k < V; oldcost[k] != inf() ==> cost[k] <= oldcost[k]);
requires (\forall int k; 0 <= k && k < V && k != src; cost[k] == inf() || cost[k] > 0);
requires 0 <= i && i < V-1;
requires 0 <= src && src < V;
requires 0 <= vertex && vertex < V;
requires Path(V, A, start, end, weight, src, vertex, P);
requires |P| <= i + 1;
requires oldcost[src] == 0 && cost[src] == 0 && cost[vertex] == inf();
requires (\forall int v; 0 <= v && v < V && ExPath(V, A, start, end, weight, src, v, i); oldcost[v] != inf());
requires (\forall int a; 0 <= a && a < A && oldcost[start[a]] != inf(); cost[end[a]] != inf());
ensures false;
pure bool lemma_inv12_helper1(int V, int A, seq<int> start, seq<int> end, seq<int> weight, seq<int> oldcost, seq<int> cost, int src, int i, int vertex, seq<int> P) =
  vertex != src && 0 < |P| && Path(V, A, start, end, weight, src, start[P[|P| - 1]], subseq(P, 0));
@*/

/*@
requires Graph(V, A, start, end, weight);
requires |oldcost| == V && |cost| == V;
requires (\forall int k; 0 <= k && k < V; cost[k] == inf() ==> oldcost[k] == cost[k]);
requires (\forall int k; 0 <= k && k < V; oldcost[k] == inf() ==> oldcost[k] <= cost[k]);
requires (\forall int k; 0 <= k && k < V; oldcost[k] != inf() ==> cost[k] <= oldcost[k]);
requires (\forall int k; 0 <= k && k < V && k != src; cost[k] == inf() || cost[k] > 0);
requires 0 <= i && i < V-1;
requires 0 <= src && src < V;
requires 0 <= vertex && vertex < V;
requires oldcost[src] == 0 && cost[src] == 0 && cost[vertex] == inf();
requires (\forall int v; 0 <= v && v < V && ExPath(V, A, start, end, weight, src, v, i); oldcost[v] != inf());
requires (\forall int a; 0 <= a && a < A && oldcost[start[a]] != inf(); cost[end[a]] != inf());
ensures (\forall seq<int> P; Path(V, A, start, end, weight, src, vertex, P) && |P| <= i + 1; false);
pure bool lemma_inv12_helper2(int V, int A, seq<int> start, seq<int> end, seq<int> weight, seq<int> oldcost, seq<int> cost, int src, int i, int vertex) =
  (\forall seq<int> P; Path(V, A, start, end, weight, src, vertex, P) && |P| <= i + 1; lemma_inv12_helper1(V, A, start, end, weight, oldcost, cost, src, i, vertex, P));
@*/

/*@
requires Graph(V, A, start, end, weight);
requires |oldcost| == V && |cost| == V;
requires (\forall int k; 0 <= k && k < V; cost[k] == inf() ==> oldcost[k] == cost[k]);
requires (\forall int k; 0 <= k && k < V; oldcost[k] == inf() ==> oldcost[k] <= cost[k]);
requires (\forall int k; 0 <= k && k < V; oldcost[k] != inf() ==> cost[k] <= oldcost[k]);
requires (\forall int k; 0 <= k && k < V && k != src; cost[k] == inf() || cost[k] > 0);
requires 0 <= i && i < V-1;
requires 0 <= src && src < V;
requires oldcost[src] == 0 && cost[src] == 0;
requires (\forall int v; 0 <= v && v < V && ExPath(V, A, start, end, weight, src, v, i); oldcost[v] != inf());
requires (\forall int a; 0 <= a && a < A && oldcost[start[a]] != inf(); cost[end[a]] != inf());
ensures (\forall int vertex; 0 <= vertex && vertex < V; (\forall seq<int> P; Path(V, A, start, end, weight, src, vertex, P) && |P| <= i + 1; vertex != inf()));
pure bool lemma_inv12_helper3(int V, int A, seq<int> start, seq<int> end, seq<int> weight, seq<int> oldcost, seq<int> cost, int src, int i) =
  (\forall int vertex; 0 <= vertex && vertex < V && cost[vertex] == inf(); lemma_inv12_helper2(V, A, start, end, weight, oldcost, cost, src, i, vertex));
@*/

/*@
requires Graph(V, A, start, end, weight);
requires |oldcost| == V && |cost| == V;
requires (\forall int k; 0 <= k && k < V; cost[k] == inf() ==> oldcost[k] == cost[k]);
requires (\forall int k; 0 <= k && k < V; oldcost[k] == inf() ==> oldcost[k] <= cost[k]);
requires (\forall int k; 0 <= k && k < V; oldcost[k] != inf() ==> cost[k] <= oldcost[k]);
requires (\forall int k; 0 <= k && k < V && k != src; cost[k] == inf() || cost[k] > 0);
requires 0 <= i && i < V-1;
requires 0 <= src && src < V;
requires oldcost[src] == 0 && cost[src] == 0;
requires (\forall int v; 0 <= v && v < V && ExPath(V, A, start, end, weight, src, v, i); oldcost[v] != inf());
requires (\forall int a; 0 <= a && a < A && oldcost[start[a]] != inf(); cost[end[a]] != inf());
ensures (\forall int v; 0 <= v && v < V && ExPath(V, A, start, end, weight, src, v, i + 1); cost[v] != inf());
pure bool lemma_inv12_preserved(int V, int A, seq<int> start, seq<int> end, seq<int> weight, seq<int> oldcost, seq<int> cost, int src, int i) =
   lemma_inv12_helper3(V, A, start, end, weight, oldcost, cost, src, i);
@*/

/* ** Invariant 11 */
/*@
requires Graph(V, A, start, end, weight);
requires |oldcost| == V && |cost| == V;
requires (\forall int k; 0 <= k && k < V; cost[k] == inf() ==> oldcost[k] == cost[k]);
requires (\forall int k; 0 <= k && k < V; oldcost[k] == inf() ==> oldcost[k] <= cost[k]);
requires (\forall int k; 0 <= k && k < V; oldcost[k] != inf() ==> cost[k] <= oldcost[k]);
requires (\forall int k; 0 <= k && k < V && k != src; cost[k] == inf() || cost[k] > 0);
requires 0 <= i && i < V-1;
requires 0 <= src && src < V;
requires 0 <= vertex && vertex < V;
requires Path(V, A, start, end, weight, src, vertex, P);
requires |P| <= i + 1;
requires oldcost[src] == 0 && cost[src] == 0 && cost[vertex] != inf();
requires cost(V, A, start, end, weight, P) < cost[vertex];
requires (\forall int v; 0 <= v && v < V && oldcost[v] != inf(); 
          (\forall seq<int> t; Path(V, A, start, end, weight, src, v, t) && 
           |t| <= i; oldcost[v] <= cost(V, A, start, end, weight, t))); // inv 11
requires (\forall int v; 0 <= v && v < V && ExPath(V, A, start, end, weight, src, v, i); oldcost[v] != inf()); // inv 12
requires (\forall int v; 0 <= v && v < V && oldcost[v] != inf(); ExPathEqual(V, A, start, end, weight, src, v, oldcost[v])); // inv 10
requires (\forall int a; 0 <= a && a < A && (oldcost[start[a]] != inf()) && 
           (oldcost[end[a]] == inf() || oldcost[start[a]] + weight[a] <= oldcost[end[a]]); cost[end[a]] <= oldcost[start[a]] + weight[a]);
ensures false;
pure bool lemma_inv11_helper1(int V, int A, seq<int> start, seq<int> end, seq<int> weight, seq<int> oldcost, seq<int> cost, int src, int i, int vertex, seq<int> P) =
     |P| > 0 &&   
     (P == subseq(P, 0) + seq<int>{P[|P| - 1]}) &&
     lemma_cost_path(V, A, start, end, weight, src, start[P[|P| - 1]], vertex, subseq(P, 0), seq<int>{P[|P| - 1]});
@*/

/*@
requires Graph(V, A, start, end, weight);
requires |oldcost| == V && |cost| == V;
requires (\forall int k; 0 <= k && k < V; cost[k] == inf() ==> oldcost[k] == cost[k]);
requires (\forall int k; 0 <= k && k < V; oldcost[k] == inf() ==> oldcost[k] <= cost[k]);
requires (\forall int k; 0 <= k && k < V; oldcost[k] != inf() ==> cost[k] <= oldcost[k]);
requires (\forall int k; 0 <= k && k < V && k != src; cost[k] == inf() || cost[k] > 0);
requires 0 <= i && i < V-1;
requires 0 <= src && src < V;
requires 0 <= vertex && vertex < V;
requires oldcost[src] == 0 && cost[src] == 0 && cost[vertex] != inf();
requires (\forall int v; 0 <= v && v < V && oldcost[v] != inf(); 
          (\forall seq<int> t; Path(V, A, start, end, weight, src, v, t) && 
           |t| <= i; oldcost[v] <= cost(V, A, start, end, weight, t))); // inv 11
requires (\forall int v; 0 <= v && v < V && ExPath(V, A, start, end, weight, src, v, i); oldcost[v] != inf()); // inv 12
requires (\forall int v; 0 <= v && v < V && oldcost[v] != inf(); ExPathEqual(V, A, start, end, weight, src, v, oldcost[v])); // inv 10
requires (\forall int a; 0 <= a && a < A && (oldcost[start[a]] != inf()) && 
           (oldcost[end[a]] == inf() || oldcost[start[a]] + weight[a] <= oldcost[end[a]]); cost[end[a]] <= oldcost[start[a]] + weight[a]);
ensures (\forall seq<int> P; Path(V, A, start, end, weight, src, vertex, P) && |P| <= i + 1 && cost(V, A, start, end, weight, P) < cost[vertex]; false);
pure bool lemma_inv11_helper2(int V, int A, seq<int> start, seq<int> end, seq<int> weight, seq<int> oldcost, seq<int> cost, int src, int i, int vertex) =
  (\forall seq<int> P; Path(V, A, start, end, weight, src, vertex, P) && |P| <= i + 1 && cost(V, A, start, end, weight, P) < cost[vertex];
    lemma_inv11_helper1(V, A, start, end, weight, oldcost, cost, src, i, vertex, P));
@*/

/*@
requires Graph(V, A, start, end, weight);
requires |oldcost| == V && |cost| == V;
requires (\forall int k; 0 <= k && k < V; cost[k] == inf() ==> oldcost[k] == cost[k]);
requires (\forall int k; 0 <= k && k < V; oldcost[k] == inf() ==> oldcost[k] <= cost[k]);
requires (\forall int k; 0 <= k && k < V; oldcost[k] != inf() ==> cost[k] <= oldcost[k]);
requires (\forall int k; 0 <= k && k < V && k != src; cost[k] == inf() || cost[k] > 0);
requires 0 <= i && i < V-1;
requires 0 <= src && src < V;
requires oldcost[src] == 0 && cost[src] == 0;
requires (\forall int v; 0 <= v && v < V && oldcost[v] != inf(); 
          (\forall seq<int> t; Path(V, A, start, end, weight, src, v, t) && 
           |t| <= i; oldcost[v] <= cost(V, A, start, end, weight, t))); // inv 11
requires (\forall int v; 0 <= v && v < V && ExPath(V, A, start, end, weight, src, v, i); oldcost[v] != inf()); // inv 12
requires (\forall int v; 0 <= v && v < V && oldcost[v] != inf(); ExPathEqual(V, A, start, end, weight, src, v, oldcost[v])); // inv 10
requires (\forall int a; 0 <= a && a < A && (oldcost[start[a]] != inf()) && 
           (oldcost[end[a]] == inf() || oldcost[start[a]] + weight[a] <= oldcost[end[a]]); cost[end[a]] <= oldcost[start[a]] + weight[a]);
ensures (\forall int vertex; 0 <= vertex && vertex < V && cost[vertex] != inf();
            (\forall seq<int> P; Path(V, A, start, end, weight, src, vertex, P) && |P| <= i + 1 && cost(V, A, start, end, weight, P) < cost[vertex]; vertex != inf()));
pure bool lemma_inv11_helper3(int V, int A, seq<int> start, seq<int> end, seq<int> weight, seq<int> oldcost, seq<int> cost, int src, int i) =
  (\forall int vertex; 0 <= vertex && vertex < V && cost[vertex] != inf(); lemma_inv11_helper2(V, A, start, end, weight, oldcost, cost, src, i, vertex));
@*/

/*@
requires Graph(V, A, start, end, weight);
requires |oldcost| == V && |cost| == V;
requires (\forall int k; 0 <= k && k < V; cost[k] == inf() ==> oldcost[k] == cost[k]);
requires (\forall int k; 0 <= k && k < V; oldcost[k] == inf() ==> oldcost[k] <= cost[k]);
requires (\forall int k; 0 <= k && k < V; oldcost[k] != inf() ==> cost[k] <= oldcost[k]);
requires (\forall int k; 0 <= k && k < V && k != src; cost[k] == inf() || cost[k] > 0);
requires 0 <= i && i < V-1;
requires 0 <= src && src < V;
requires oldcost[src] == 0 && cost[src] == 0;
requires (\forall int v; 0 <= v && v < V && oldcost[v] != inf(); 
          (\forall seq<int> t; Path(V, A, start, end, weight, src, v, t) && 
           |t| <= i; oldcost[v] <= cost(V, A, start, end, weight, t))); // inv 11
requires (\forall int v; 0 <= v && v < V && ExPath(V, A, start, end, weight, src, v, i); oldcost[v] != inf()); // inv 12
requires (\forall int v; 0 <= v && v < V && oldcost[v] != inf(); ExPathEqual(V, A, start, end, weight, src, v, oldcost[v])); // inv 10
requires (\forall int a; 0 <= a && a < A && (oldcost[start[a]] != inf()) && 
           (oldcost[end[a]] == inf() || oldcost[start[a]] + weight[a] <= oldcost[end[a]]); cost[end[a]] <= oldcost[start[a]] + weight[a]);
ensures (\forall int vertex; 0 <= vertex && vertex < V && cost[vertex] != inf();
            (\forall seq<int> P; Path(V, A, start, end, weight, src, vertex, P) && |P| <= i + 1; cost(V, A, start, end, weight, P) >= cost[vertex]));
pure bool lemma_inv11_preserved(int V, int A, seq<int> start, seq<int> end, seq<int> weight, seq<int> oldcost, seq<int> cost, int src, int i) = 
  lemma_inv11_helper3(V, A, start, end, weight, oldcost, cost, src, i);
@*/

//////////////////////////////////////////////////////////////////////////////////Lemma1_simple_path
/*@
requires bound >= 0;
pure bool inbounds(seq<int> xs, int bound) = (\forall int i; i >= 0 && i < |xs|; xs[i] < bound);
@*/

/*@
pure bool nodupl(seq<int> xs) = 
(\forall int i; i >= 0 && i < |xs|; (\forall int j; j > i && j < |xs|; xs[i] != xs[j]));
@*/

/**
 * Gives the set containing all elements ranging from 0 up to (and including) `max`
 */
/*@
requires max >= 0;
ensures (\forall int v; v >= 0 && v <= max; (v \in \result));
ensures (\forall int v; (v \in \result); v >= 0 && v <= max);
ensures |\result| == max + 1;
pure set<int> rangeset(int max) = max > 0 ? set<int>{max} + rangeset(max-1) : set<int>{0};
@*/

/**
 * Converts `xs` to a set.
 */ 
/*@
ensures (\forall int v; (v \in \result); (v \in xs));
ensures (\forall int v; (v \in xs); (v \in \result));
pure set<int> seq2set(seq<int> xs) = |xs| > 0 ? set<int>{xs[0]} + seq2set(tail(xs)) : set<int>{};
@*/

/**
 * If all elements in `xs` are unique, then `xs` has the same size as `seq2set(xs)`.
 */
/*@
requires nodupl(xs);
ensures \result && |xs| == |seq2set(xs)|;
pure bool seq2set_nodupl(seq<int> xs) = |xs| > 0 ==> seq2set_nodupl(tail(xs));
@*/

/**
 * If all elements in `xs` are unique and within 0..`bound`,
 * then `xs` contains at most `bound` elements.
 */
/*@
requires 0 < bound;
requires inbounds(xs, bound);
requires nodupl(xs);
ensures \result;
pure bool seq_bounded_size(seq<int> xs, int bound) = 
  |seq2set(xs) - rangeset(bound - 1)| == |seq2set(xs)| - |seq2set(xs) * rangeset(bound - 1)| && seq2set_nodupl(xs);
@*/

/**
 * Gives the list of all vertices that are on `P`.
 * Since `P` is simple, all these vertices must be unique.
 */
/*@
requires Graph(V, A, start, end, weight);
requires SimplePath(V, A, start, end, weight, x, y, P);
ensures |\result| == |P| + 1;
ensures \result[0] == x;
ensures (\forall int i; 0 < i && i < |\result|; \result[i] == end[P[i - 1]]);
ensures inbounds(\result, V);
ensures nodupl(\result);
pure seq<int> collect(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x, int y, seq<int> P) =
  seq<int>{x} + collect_helper(V, A, start, end, weight, x, y, P);
@*/

/**
 * Auxiliary helper function for `collect`.
 */
/*@
requires Graph(V, A, start, end, weight);
requires SimplePath(V, A, start, end, weight, x, y, P);
ensures |\result| == |P|;
ensures (\forall int i; 0 <= i && i < |\result|; \result[i] == end[P[i]]);
ensures inbounds(\result, V);
ensures nodupl(\result);
pure seq<int> collect_helper(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x, int y, seq<int> P) =
  0 < |P| ? seq<int>{end[P[0]]} + collect_helper(V, A, start, end, weight, end[P[0]], y, tail(P)) : seq<int>{};
@*/

/*@
requires Graph(V, A, start, end, weight);
requires SimplePath(V, A, start, end, weight, x, y, P);
ensures \result && |P| < V;  
pure bool simple_path_lemma1_helper(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x, int y, seq<int> P) =
  seq_bounded_size(collect(V, A, start, end, weight, x, y, P), V);
@*/

/*@
requires Graph(V, A, start, end, weight);
ensures (\forall seq<int> P; SimplePath(V, A, start, end, weight, x, y, P); |P| < V);
pure bool simple_path_lemma1(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x, int y) =  
  (\forall seq<int> P; SimplePath(V, A, start, end, weight, x, y, P); simple_path_lemma1_helper(V, A, start, end, weight, x, y, P));
@*/

/*@
requires Graph(V, A, start, end, weight);
ensures \result && (\forall int v; v >= 0 && v < V; (\forall seq<int> P; SimplePath(V, A, start, end, weight, x, v, P); |P| < V));
pure bool simple_path_lemma1_preserved(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x) = 
  (\forall int v; v >= 0 && v < V; (\forall seq<int> P; SimplePath(V, A, start, end, weight, x, v, P); simple_path_lemma1(V, A, start, end, weight, x, v)) ); 
@*/

//////////////////////////////////////////////////////////////////////////////////Lemma2&3&4_simple_path
/**
  * Given any simple (x,y)-path `P` and simple (y,z)-path `Q`
  * for which it holds that `P[1..] ++ Q` is also a simple path (see fourth requires clause),
  * this function constructs and returns a simple (x,z)-path.
  */
/*@
requires Graph(V, A, start, end, weight);
requires SimplePath(V, A, start, end, weight, x, y, P);
requires SimplePath(V, A, start, end, weight, y, z, Q);
requires (\forall int i; i > 0 && i < |P|; 
          (\forall int j; j >= 0 && j < |Q|; start[P[i]] != end[Q[j]]));
requires (\forall int i; i >= 0 && i < |P|; end[P[i]] != x);
ensures SimplePath(V, A, start, end, weight, x, z, \result);
ensures |\result| <= |P| + |Q|;
ensures cost(V, A, start, end, weight, \result) <= cost(V, A, start, end, weight, P) + cost(V, A, start, end, weight, Q);
pure seq<int> trim(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x, int y, int z, seq<int> P, seq<int> Q) =
    0 < |Q| ? (end[Q[0]] == x ?
               trim(V, A, start, end, weight, x, x, z, seq<int>{}, tail(Q)) :
               assertSeq(trim(V, A, start, end, weight, x, end[Q[0]], z, P+seq<int>{Q[0]}, tail(Q)), 
               (lemma_cost_app(V, A, start, end, weight, P+seq<int>{Q[0]}, tail(Q)) &&
               lemma_app_append_right(P, Q) && 
               lemma_cost_app(V, A, start, end, weight, P, Q)))) 
            : P;
@*/  

/**
 * Given any simple (y,z)-path `P` and (x,y)-arc `a`,
 * this function constructs and returns a simple (x,z)-path.
 */
/*@
requires Graph(V, A, start, end, weight);
requires SimplePath(V, A, start, end, weight, y, z, P);
requires 0 <= a && a < A && start[a] == x && end[a] == y;
ensures SimplePath(V, A, start, end, weight, x, z, \result);
ensures |\result| <= |P| + 1; 
ensures cost(V, A, start, end, weight, \result) <= weight[a] + cost(V, A, start, end, weight, P);
pure  seq<int> extend(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x, int y, int z, int a, seq<int> P) =
   trim(V, A, start, end, weight, x, y, z, seq<int>{a}, P);
@*/

/**
 * Converts any given (x,y)-path `P` to a simple (x,y)-path.
 * The other Half part of lemma 2
 * And lemma 3 And lemma 4
 */
/*@  
requires Graph(V, A, start, end, weight);
requires Path(V, A, start, end, weight, x, y, P);
ensures SimplePath(V, A, start, end, weight, x, y, \result);
ensures |\result| <= |P|;
ensures cost(V, A, start, end, weight, \result) <= cost(V, A, start, end, weight, P);
pure seq<int> convert(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x, int y, seq<int> P) =
  |P| > 0 ? extend(V, A, start, end, weight, x, end[P[0]], y, P[0], convert(V, A, start, end, weight, end[P[0]], y, tail(P)))
  : P;
@*/

/*@ 
requires Graph(V, A, start, end, weight);
requires SimplePath(V, A, start, end, weight, x, y, P);
ensures \result && Path(V, A, start, end, weight, x, y, P);    
pure bool simple_path_Part_1_lemma2_helper(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x, int y, seq<int> P) = true;
@*/

/*@
requires Graph(V, A, start, end, weight);
ensures (\forall seq<int> P; SimplePath(V, A, start, end, weight, x, y, P); Path(V, A, start, end, weight, x, y, P));
pure bool simple_path_Part_1_lemma2(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x, int y) =
(\forall seq<int> P; SimplePath(V, A, start, end, weight, x, y, P); simple_path_Part_1_lemma2_helper(V, A, start, end, weight, x, y, P));
@*/


/**
 * Every simple path is a path (trivial).
 */
/*@
requires Graph(V, A, start, end, weight);
ensures \result && (\forall int v; v >= 0 && v < V; 
         (\forall seq<int> P; SimplePath(V, A, start, end, weight, x, v, P); Path(V, A, start, end, weight, x, v, P)));
pure bool simple_path_Part_1_lemma2_preserved(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x) =
   (\forall int v; v >= 0 && v < V; simple_path_Part_1_lemma2(V, A, start, end, weight, x, v));
@*/

/**
 * For every path convert(path) is a path.
 * For every path convert(path) is a simple path.
 */
/*@
requires Graph(V, A, start, end, weight);
ensures (\forall int v; v >= 0 && v < V; 
         (\forall seq<int> P; Path(V, A, start, end, weight, x, v, P); Path(V, A, start, end, weight, x, v, convert(V, A, start, end, weight, x, v, P))));
ensures \result && (\forall int v; v >= 0 && v < V; 
         (\forall seq<int> P; Path(V, A, start, end, weight, x, v, P); SimplePath(V, A, start, end, weight, x, v, convert(V, A, start, end, weight, x, v, P))));
pure bool simple_path_Part_2_lemma2_preserved(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x) = true;
@*/

/**
 * For every path |convert(path)| <= |path|.
 */
/*@
requires Graph(V, A, start, end, weight);
ensures \result && (\forall int v; v >= 0 && v < V; 
         (\forall seq<int> P; Path(V, A, start, end, weight, x, v, P); 
           |convert(V, A, start, end, weight, x, v, P)| <= |P|));
pure bool simple_path_lemma3_preserved(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x) = true;
@*/

/**
 * For every path cost(convert(path)) <= cost(path).
 */
/*@
requires Graph(V, A, start, end, weight);
ensures \result && (\forall int v; v >= 0 && v < V; 
         (\forall seq<int> P; Path(V, A, start, end, weight, x, v, P); 
            cost(V, A, start, end, weight, convert(V, A, start, end, weight, x, v, P)) <= cost(V, A, start, end, weight, P)));
pure bool simple_path_cost_lemma4(int V, int A, seq<int> start, seq<int> end, seq<int> weight, int x) = true;
@*/


////////////////////////////////////////////////////////////////////////////////
//compare
////////////////////////////////////////////////////////////////////////////////  
  /*@
  requires start_cost == inf() || start_cost >= 0;
  requires end_cost == inf() || end_cost >= 0;
  requires weight > 0;
  ensures (start_cost != inf()) && (end_cost == inf() || start_cost + weight <= end_cost) ==> true;
  ensures true ==> (start_cost != inf()) && (end_cost == inf() || start_cost + weight <= end_cost);
  ensures (start_cost == inf()) || (end_cost != inf() && start_cost + weight > end_cost) ==> false;
  ensures false ==> (start_cost == inf()) || (end_cost != inf() && start_cost + weight > end_cost);
  ensures (true && end_cost != inf()) ==> start_cost + weight <= end_cost;
  pure bool compare(int start_cost, int weight, int end_cost) = 
    (start_cost != inf() && (end_cost == inf() || start_cost+weight <= end_cost)) ? true : false;
  @*/
////////////////////////////////////////////////////////////////////////////////
//Kernel
////////////////////////////////////////////////////////////////////////////////
/*@
  yields seq<int> contrib;
  given seq<int> cost_seq;
  given seq<int> oldcost_seq;
  given seq<int> start_seq;
  given seq<int> end_seq;
  given seq<int> weight_seq; 

  context_everywhere V == 1024 && A == 10 * V;
  context_everywhere opencl_gsize == V;
  context_everywhere 10 == opencl_gcount;
  context_everywhere source >= 0 && source < V;
  context_everywhere counter >= 0 && counter < V-1;
  context_everywhere g_start.length == A && g_end.length == A && g_weight.length == A && g_cost.length == V;
  context_everywhere |start_seq| == A && |end_seq| == A && |weight_seq| == A && |cost_seq| == V && |contrib| == A;

  kernel_invariant (\forall* int i; 0 <= i && i < A; Perm(g_start[i], 1\4));
  kernel_invariant (\forall* int i; 0 <= i && i < A; Perm(g_end[i], 1\4));
  kernel_invariant (\forall int i; 0 <= i && i < A; g_start[i] >= 0 && g_start[i] < V && g_end[i] >= 0 && g_end[i] < V);
  kernel_invariant (\forall int i; 0 <= i && i < A; g_start[i] != g_end[i]);
  kernel_invariant (\forall int i; 0 <= i && i < A; (\forall int j; 0 <= j && j < A && i != j; g_start[i] == g_start[j] ==> g_end[i] != g_end[j]));
  
  kernel_invariant (\forall* int i; 0 <= i && i < A; Perm(g_weight[i], 1\4));
  kernel_invariant (\forall int i; 0 <= i && i < A; weight[i] > 0);
  
  kernel_invariant (\forall* int i; 0 <= i && i < V; Perm(g_cost[i], write));

  kernel_invariant (\forall int i; 0 <= i && i < A; start[i] == start_seq[i]);
  kernel_invariant (\forall int i; 0 <= i && i < A; end[i] == end_seq[i]);
  kernel_invariant (\forall int i; 0 <= i && i < A; weight[i] == weight_seq[i]); 
  kernel_invariant (\forall int i; 0 <= i && i < A; contrib1[i] == 0) ==> (\forall int i; 0 <= i && i < V; oldcost_seq[i] == cost_seq[i]);
  kernel_invariant (\forall int i; 0 <= i && i < V; cost_seq[i] == inf() ==> oldcost_seq[i] == cost_seq[i]);
  kernel_invariant (\forall int i; 0 <= i && i < V; oldcost_seq[i] == inf() ==> oldcost_seq[i] <= cost_seq[i]);
  kernel_invariant (\forall int i; 0 <= i && i < V; oldcost_seq[i] != inf() ==> cost_seq[i] <= oldcost_seq[i]); 
  kernel_invariant (\forall int i; 0 <= i && i < V && i != source; oldcost_seq[i] == inf() || oldcost_seq[i] > 0); 
  kernel_invariant oldcost_seq[source] == 0;
  kernel_invariant (\forall int i; 0 <= i && i < V && i != source; cost_seq[i] == inf() || cost_seq[i] > 0);
  kernel_invariant cost_seq[source] == 0;
  kernel_invariant (\forall int i; 0 <= i && i < V; cost[i] == cost_seq[i]);
  kernel_invariant (\forall int i; 0 <= i && i < A && contrib[i] == 1 && oldcost_seq[start_seq[i]] != inf(); cost_seq[end_seq[i]] != inf());
  kernel_invariant (\forall int a; 0 <= a && a < A && contrib1[a] == 1 && (oldcost_seq[start[a]] != inf()) && 
                    (oldcost_seq[end[a]] == inf() || oldcost_seq[start[a]] + weight_seq[a] <= oldcost_seq[end[a]]); cost[end[a]] <= oldcost_seq[start[a]] + weight_seq[a]);
  kernel_invariant (\forall int i; 0 <= i && i < A && cost_seq[start[i]] != inf(); ExPathEqual(V, A, start_seq, end_seq, weight_seq, source, start[i], cost_seq[start[i]])); // inv 10
  kernel_invariant (\forall int i; 0 <= i && i < A && cost_seq[end[i]] != inf(); ExPathEqual(V, A, start_seq, end_seq, weight_seq, source, end[i], cost_seq[end[i]])); // inv 10 
  kernel_invariant (\forall int i; 0 <= i && i < V && cost_seq[i] != inf(); ExPathEqual(V, A, start_seq, end_seq, weight_seq, source, i, cost_seq[i])); // inv 10
  kernel_invariant (\forall int v; 0 <= v && v < V && oldcost_seq[v] != inf(); ExPathEqual(V, A, start_seq, end_seq, weight_seq, source, v, oldcost_seq[v])); // inv 10
  kernel_invariant (\forall int vertex; 0 <= vertex && vertex < V && oldcost_seq[vertex] != inf(); 
                    (\forall seq<int> P; Path(V, A, start_seq, end_seq, weight_seq, source, vertex, P) && |P| <= counter[0]; cost(V, A, start_seq, end_seq, weight_seq, P) >= oldcost_seq[vertex])); // inv 11
  kernel_invariant (\forall int v; 0 <= v && v < V && ExPath(V, A, start_seq, end_seq, weight_seq, source, v, counter[0]); oldcost_seq[v] != inf()); // inv 12
  

  requires \pointer_index(g_start, \gtid, 1\4);
  requires \pointer_index(g_end, \gtid, 1\4);
  requires \pointer_index(g_weight, \gtid, 1\4);
  ensures contrib[\gtid] == 1;
@*/
__global__ void CUDAKernel(int* g_start, int* g_end, int* g_weight, int* g_cost, int V, int A, int counter, int source)
{
  int tid = blockIdx.x * V + threadIdx.x;
  //@ assert tid == \gtid;

  atomicRelax(g_cost+g_end[tid], g_weight[tid], g_cost[g_start[tid]]) 
    /*@ then {
        cost_seq[end[tid]] = (compare(oldcost_seq[start_seq[tid]], weight_seq[tid], oldcost_seq[end_seq[tid]]) ?  (cost[start[tid]] + weight[tid]) : cost_seq[end[tid]]);
                compare(oldcost_seq[start_seq[tid]], weight_seq[tid], oldcost_seq[end_seq[tid]]) ? lemma_one_path(V, A, start_seq, end_seq, weight_seq) : true;
                compare(oldcost_seq[start_seq[tid]], weight_seq[tid], oldcost_seq[end_seq[tid]]) ? ExPathEqual(V, A, start_seq, end_seq, weight_seq, start[tid], end[tid], weight[tid]) : true;
                compare(oldcost_seq[start_seq[tid]], weight_seq[tid], oldcost_seq[end_seq[tid]]) ? lemma_expath_trans(V, A, start_seq, end_seq, weight_seq, source, start[tid], end[tid], cost_seq[start[tid]], weight[tid]) : true;
                compare(oldcost_seq[start_seq[tid]], weight_seq[tid], oldcost_seq[end_seq[tid]]) ? (cost[end[tid]] == cost[start[tid]] + weight[tid]) : true;
                compare(oldcost_seq[start_seq[tid]], weight_seq[tid], oldcost_seq[end_seq[tid]]) ? ExPathEqual(V, A, start_seq, end_seq, weight_seq, source, end[tid], cost_seq[start[tid]]+weight[tid]) : true;
                contrib[tid] = 1;
    } @*/;
   
    /* I would like to have such an atomic (block) which I encoded it as "atomicRelax"
  Atomic{
  
    if(atomicCompare(g_cost[g_end[tid]], g_weight[tid], g_cost[g_start[tid]])){
	
	    g_cost[g_end[tid]] = g_cost[g_start[tid]] + g_weight[tid];

	    //@ ghost cost_seq = cost_seq[end[tid] -> (cost[start[tid]] + weight[tid])];
      //@ ghost lemma_one_path(V, A, start_seq, end_seq, weight_seq);
      //@ assert ExPathEqual(V, A, start_seq, end_seq, weight_seq, start[tid], end[tid], weight[tid]); 
      //@ assert lemma_expath_trans(V, A, start_seq, end_seq, weight_seq, source, start[tid], end[tid], cost_seq[start[tid]], weight[tid]);
      //@ assert cost[end[tid]] == cost[start[tid]] + weight[tid];
      //@ assert ExPathEqual(V, A, start_seq, end_seq, weight_seq, source, end[tid], cost_seq[start[tid]]+weight[tid]);  
 
        
    }

   }//@ then {contrib[tid] = 1;} @//;	
    
   requires start_cost == inf() || start_cost >= 0;
   requires end_cost == inf() || end_cost >= 0;
   requires weight > 0;
   ensures (start_cost != inf()) && (end_cost == inf() || start_cost + weight <= end_cost) ==> true;
   ensures true ==> (start_cost != inf()) && (end_cost == inf() || start_cost + weight <= end_cost);
   ensures (start_cost == inf()) || (end_cost != inf() && start_cost + weight > end_cost) ==> false;
   ensures false ==> (start_cost == inf()) || (end_cost != inf() && start_cost + weight > end_cost);
   ensures (true && end_cost != inf()) ==> start_cost + weight <= end_cost;
   boolean atomicCompare(int start_cost, int weight, int end_cost) =
    (start_cost != inf()) && (end_cost == inf() || start_cost + weight <= end_cost) ?
          true
    :
          false;
    */  


}

////////////////////////////////////////////////////////////////////////////////
//@ ensures \pointer(\result, N, write);
int *vercorsMallocInt(int N);
void vercorsFreeInt(int *ar);
//@ ensures \pointer(\result, N, write);
int *vercorsCudaMallocInt(int N);
void vercorsCudaFreeInt(int *addr);
void vercorsCudaMemcpyInt(int *tgt, int *src, int N, int direction); 

////////////////////////////////////////////////////////////////////////////////
// Main Program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv) 
{
	
	int V = 1024; // no. of vertices
	int A = 10 * V; // no. of arcs
    	
	// allocate host memory
  int* host_start = vercorsMallocInt(A);
  int* host_end = vercorsMallocInt(A);
  int* host_weight = vercorsMallocInt(A);
  int* host_cost = vercorsMallocInt(V);
  
  int source = V/2;
  
  // these assumptions come from the input graph file which we change it into assertions 
  //@ assume (\forall int i; 0 <= i && i < A; host_start[i] >= 0 && host_start[i] < V);
  //@ assume (\forall int i; 0 <= i && i < A; host_end[i] >= 0 && host_end[i] < V);
  //@ assume (\forall int i; 0 <= i && i < A; host_start[i] != host_end[i]); // No self-loop
  
  //@ assume (\forall int i; 0 <= i && i < A; host_weight[i] > 0);
  
  //@ assume (\forall int i; 0 <= i && i < A; (\forall int j; 0 <= j && j < A && i != j; host_start[i] == host_start[j] ==> host_end[i] != host_end[j])); 
     
  for(unsigned int i = 0; i < V; i++) 
  {
    host_cost[i] = -1;
  }
  host_cost[source] = 0;
  
  //@ assert (\forall int i; 0 <= i && i < V && i != source; host_cost[i] == inf());
  
  //@ assert host_cost[source] == 0;

  //Copy the arrays to device memory
  int* device_start;
  device_start = vercorsCudaMallocInt(A);
  vercorsCudaMemcpyInt( device_start, host_start, A, hipMemcpyHostToDevice) ;

  //Copy the arrays to device memory
  int* device_end;
  device_end = vercorsCudaMallocInt(A);
  vercorsCudaMemcpyInt( device_end, host_end, A, hipMemcpyHostToDevice) ;
  
  //Copy the arrays to device memory
  int* device_weight;
  device_weight = vercorsCudaMallocInt(A);
  vercorsCudaMemcpyInt( device_weight, host_weight, A, hipMemcpyHostToDevice) ;


  //Copy the arrays to device memory
  int* device_cost;
  device_cost = vercorsCudaMallocInt(V);
  vercorsCudaMemcpyInt( device_cost, host_cost, V, hipMemcpyHostToDevice) ;
  
    
  //setup execution parameters
	int num_of_blocks = 10;
	int num_of_threads_per_block = V;
  
  //dim3  grid( num_of_blocks, 1, 1); //grid has three parameters to indicate the dimensions. Here we have one dimensional grid (of blocks). It can be one, two or three dimensions.
  //dim3  threads( num_of_threads_per_block, 1, 1); //threads indicates the dimensions of one block. Here each block has one dimension (of threads). It can be one, two or three dimensions.
  
  int counter = 0;

  //@ ghost seq<int> contrib;
  //@ assume (\forall int i; i >= 0 && i < A; contrib[i] == 0);
  
  //@ ghost seq<int> cost_seq ;
  //@ assume |cost_seq| == V && (\forall int i; i >= 0 && i < V; cost_seq[i] == host_cost[i]);
  
  //@ ghost seq<int> oldcost_seq ; 
  //@ assume |oldcost_seq| == V && (\forall int i; i >= 0 && i < V; oldcost_seq[i] == host_cost[i]);
  
  //@ ghost seq<int> start_seq ; 
  //@ assume |start_seq| == A && (\forall int i; i >= 0 && i < A; start_seq[i] == host_start[i]);
  
  //@ ghost seq<int> end_seq ;
  //@ assume |end_seq| == A && (\forall int i; i >= 0 && i < A; end_seq[i] == host_end[i]);
  
  //@ ghost seq<int> weight_seq  ;
  //@ assume |weight_seq| == A && (\forall int i; i >= 0 && i < A; weight_seq[i] == host_weight[i]);

  



  //@ loop_invariant V > 0 && A > 0;
  //@ loop_invariant source >= 0 && source < V;
  //@ loop_invariant counter.length == 1;
  //@ loop_invariant Perm(counter[0], write);
  //@ loop_invariant counter[0] >= 0 && counter[0] <= V - 1;
  //@ loop_invariant start != null;
  //@ loop_invariant weight != null;
  //@ loop_invariant end != null;
  //@ loop_invariant cost != null;
  //@ loop_invariant start.length == A;
  //@ loop_invariant weight.length == A;
  //@ loop_invariant end.length == A;
  //@ loop_invariant cost.length == V;
  //@ loop_invariant Perm(start_seq, read);
  //@ loop_invariant |start_seq| == A;
  //@ loop_invariant Perm(weight_seq, read);
  //@ loop_invariant |weight_seq| == A;
  //@ loop_invariant Perm(end_seq, read);
  //@ loop_invariant |end_seq| == A;
  //@ loop_invariant Perm(cost_seq, 1);
  //@ loop_invariant Perm(oldcost_seq, 1);
  //@ loop_invariant |cost_seq| == V;
  //@ loop_invariant |oldcost_seq| == V;
  //@ loop_invariant (\forall int i; 0 <= i && i < V; oldcost_seq[i] == cost_seq[i]);
  //@ loop_invariant contrib1 != null;
  //@ loop_invariant contrib1.length == A;
  //@ loop_invariant (\forall* int i; 0 <= i && i < A; Perm(contrib1[i], write));
  //@ loop_invariant (\forall int i; 0 <= i && i < A; contrib1[i] == 0);
  //@ loop_invariant (\forall* int i; 0 <= i && i < A; Perm(start[i], read));
  //@ loop_invariant (\forall int i; 0 <= i && i < A; start[i] >= 0 && start[i] < V);
  //@ loop_invariant (\forall* int i; 0 <= i && i < A; Perm(end[i], read));
  //@ loop_invariant (\forall int i; 0 <= i && i < A; end[i] >= 0 && end[i] < V);
  //@ loop_invariant (\forall int i; 0 <= i && i < A; start[i] != end[i]);
  //@ loop_invariant (\forall int i; 0 <= i && i < A; (\forall int j; 0 <= j && j < A && i != j; start[i] == start[j] ==> end[i] != end[j]));
  //@ loop_invariant (\forall int i; 0 <= i && i < A; start[i] == start_seq[i]);
  //@ loop_invariant (\forall int i; 0 <= i && i < A; end[i] == end_seq[i]);
  //@ loop_invariant (\forall* int i; 0 <= i && i < A; Perm(weight[i], read));
  //@ loop_invariant (\forall int i; 0 <= i && i < A; weight[i] == weight_seq[i]);
  //@ loop_invariant (\forall int i; 0 <= i && i < A; weight[i] > 0);
  //@ loop_invariant (\forall* int i; 0 <= i && i < V; Perm(cost[i], write));
  //@ loop_invariant (\forall int i; 0 <= i && i < V && i != source; cost_seq[i] == inf() || cost_seq[i] > 0);
  //@ loop_invariant cost[source] == 0;
  //@ loop_invariant (\forall int i; 0 <= i && i < V; cost[i] == cost_seq[i]);
  //@ loop_invariant (\forall int i; 0 <= i && i < V && cost_seq[i] != inf(); ExPathEqual(V, A, start_seq, end_seq, weight_seq, source, i, cost_seq[i]));
  //@ loop_invariant (\forall int vertex; 0 <= vertex && vertex < V && cost_seq[vertex] != inf(); (\forall seq<int> P; Path(V, A, start_seq, end_seq, weight_seq, source, vertex, P) && |P| <= counter[0]; cost_seq[vertex] <= cost(V, A, start_seq, end_seq, weight_seq, P)));
  //@ loop_invariant (\forall int v; 0 <= v && v < V && ExPath(V, A, start_seq, end_seq, weight_seq, source, v, counter[0]); cost_seq[v] != inf());
  while(counter < V-1)
  {  
	  //Kernel launch
    CUDAKernel<<< /*grid*/num_of_blocks, /*threads*/num_of_threads_per_block/*, 0*/ >>>(device_start, device_end, device_weight, device_cost, V, A, counter, source)
        /*@ with { contrib=contrib; cost_seq=cost_seq; oldcost_seq=oldcost_seq; start_seq=start_seq; end_seq=end_seq; weight_seq=weight_seq; } @*/;
    
    //@ ghost contrib[A] = 0 ;
    //@ assume (\forall int i; i >= 0 && i < A; contrib[i] == 0);
    
    //@ assert lemma_inv12_preserved(V, A, start_seq, end_seq, weight_seq, oldcost_seq, cost_seq, source, counter);
    
    //@ assert lemma_inv11_preserved(V, A, start_seq, end_seq, weight_seq, oldcost_seq, cost_seq, source, counter);
    
    //@ ghost oldcost_seq = cost_seq ;
    //@ assume (\forall int i; i >= 0 && i < V; oldcost_seq[i] == cost_seq[i]);

    counter = counter + 1;
  }
  
  //Lemma 1
  //@ assert simple_path_lemma1_preserved(V, A, start_seq, end_seq, weight_seq, source);
  //@ assert (\forall int v; v >= 0 && v < V; (\forall seq<int> P; SimplePath(V, A, start_seq, end_seq, weight_seq, source, v, P); |P| < V));
  
  //Lemma 2
  //@ assert simple_path_Part_1_lemma2_preserved(V, A, start_seq, end_seq, weight_seq, source);
  //@ assert (\forall int v; v >= 0 && v < V; (\forall seq<int> P; SimplePath(V, A, start_seq, end_seq, weight_seq, source, v, P); Path(V, A, start_seq, end_seq, weight_seq, source, v, P)));
  //@ assert simple_path_Part_2_lemma2_preserved(V, A, start_seq, end_seq, weight_seq, source);
  //@assert (\forall int v; v >= 0 && v < V; (\forall seq<int> P; Path(V, A, start_seq, end_seq, weight_seq, source, v, P); Path(V, A, start_seq, end_seq, weight_seq, source, v, convert(V, A, start_seq, end_seq, weight_seq, source, v, P))));
  //@ assert (\forall int v; v >= 0 && v < V; (\forall seq<int> P; Path(V, A, start_seq, end_seq, weight_seq, source, v, P); SimplePath(V, A, start_seq, end_seq, weight_seq, source, v, convert(V, A, start_seq, end_seq, weight_seq, source, v, P))));
  
  //Lemma 3
  //@ assert simple_path_lemma3_preserved(V, A, start_seq, end_seq, weight_seq, source);
  //@ assert (\forall int v; v >= 0 && v < V;(\forall seq<int> P; Path(V, A, start_seq, end_seq, weight_seq, source, v, P); |convert(V, A, start_seq, end_seq, weight_seq, source, v, P)| <= |P|));
                     
  //Lemma 4
  //@ assert simple_path_cost_lemma4(V, A, start_seq, end_seq, weight_seq, source);
  //@ assert (\forall int v; v >= 0 && v < V; (\forall seq<int> P; Path(V, A, start_seq, end_seq, weight_seq, source, v, P); cost(V, A, start_seq, end_seq, weight_seq, convert(V, A, start_seq, end_seq, weight_seq, source, v, P)) <= cost(V, A, start_seq, end_seq, weight_seq, P)));
           
                    
  
  //@ assert (\forall int vertex; 0 <= vertex && vertex < V && cost_seq[vertex] != inf(); (\forall seq<int> P; Path(V, A, start_seq, end_seq, weight_seq, source, vertex, P); cost_seq[vertex] <= cost(V, A, start_seq, end_seq, weight_seq, P))); // prop. 8
  
  //@ assert (\forall int v; 0 <= v && v < V && (\exists seq<int> P; Path(V, A, start_seq, end_seq, weight_seq, source, v, P); true); cost_seq[v] != inf()); // prop. 9
  
  
  // copy result from device to host
  vercorsCudaMemcpyInt(host_cost, device_cost, V, hipMemcpyDeviceToHost);	
	 
  
  // cleanup memory
  vercorsFreeInt(host_start);
  vercorsFreeInt(host_end);
  vercorsFreeInt(host_weight);
  vercorsFreeInt(host_cost);
  vercorsCudaFreeInt(device_start);
  vercorsCudaFreeInt(device_end);
  vercorsCudaFreeInt(device_weight);
  vercorsCudaFreeInt(device_cost);

	return 0;
       
}

