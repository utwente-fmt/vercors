#include "hip/hip_runtime.h"
//:: case GPGPUExampleUpdates

/* This is the realistic version of GPGPU-Example.cu, with the difference that this version uses actual CUDA functions,
 * such as hipMemcpy. Unfortunately this file cannot be verified yet as VerCors does not yet support these primitives.
 * Therefore it is not included in the test suite.
 */

/***********************************************************************************
Created by Mohsen Safari.
************************************************************************************/
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>


////////////////////////////////////////////////////////////////////////////////
//Kernel
////////////////////////////////////////////////////////////////////////////////
/*@
  given int contrib;
  kernel_invariant Perm(g_sum, 1) ** (\forall* int i; i >= 0 && i < N; Perm(contrib[i], 1\2)) ** g_sum == (\sum int i; 0 <= i && i < N; contrib[i]);
  requires Perm(g_array[\gtid], 1);
  requires (\forall* int i; i >= 0 && i < N; Perm(contrib[i], 1\2));
  requires contrib[\gtid] == 0;
  ensures contrib[\gtid] == g_array[\gtid];
  ensures (\forall* int i; i >= 0 && i < N; Perm(contrib[i], 1\2));
  ensures Perm(g_array[\gtid], 1);
@*/
__global__ void CUDAKernel(int* g_array, int* g_sum, int N)
{
  int tid = blockIdx.x * N + threadIdx.x;
  // each thread atomicaly adds its own value into g_sum
  atomicAdd(&g_sum, g_array[tid]) /*@ then {contrib[\gtid] = g_array[\gtid];} @*/;
}

////////////////////////////////////////////////////////////////////////////////
// Main Program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{

  int N = 1024; // size of the array

  // allocate host memory
  int* host_array = (int*) malloc(sizeof(int)*N);
  int host_sum = 0;
  // initalize the memory
  for(unsigned int i = 0; i < N; i++)
  {
    host_array[i] = i;
  }


  //Copy the array to device memory
  int* device_array;
  hipMalloc( (void**) &device_array, sizeof(int)*N) ;
  hipMemcpy( device_array, host_array, sizeof(int)*N, hipMemcpyHostToDevice) ;

  //Copy the int variable to device memory
  int* device_sum;
  hipMalloc((void**) &device_sum, sizeof(int));
  hipMemcpy( device_sum, host_sum, sizeof(int), hipMemcpyHostToDevice);


  //setup execution parameters
  int num_of_blocks = 1;
  int num_of_threads_per_block = N;

  //start the timer
  hipEvent_t begin, end;
  float time;
  hipEventCreate(&begin);
  hipEventCreate(&end);
  hipEventRecord(begin, 0);

  /*@ ghost int contrib[N] = 0; @*/

  //Kernel launch
  CUDAKernel<<< num_of_blocks, num_of_threads_per_block >>>(device_array, device_sum, N) /*@ with { contrib = contrib; } @*/;

  //Stop the Timer
  hipEventRecord(end, 0);
  hipEventSynchronize(end);
  hipEventElapsedTime(&time, begin, end);
  hipEventDestroy(begin);
  hipEventDestroy(end);

  // copy result from device to host
  hipMemcpy(host_sum, device_sum, sizeof(int), hipMemcpyDeviceToHost);

  //print kernel execution time
  printf( "Processing time: %f (ms)\n", time);

  //print the result
  printf( "Sum of the array values: %d \n", host_sum);

  // cleanup memory
  free(host_array);
  hipFree(device_array);

  return 0;

}
