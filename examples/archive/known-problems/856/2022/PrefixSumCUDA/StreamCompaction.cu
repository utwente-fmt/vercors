#include "hip/hip_runtime.h"
/***********************************************************************************
Created by Mohsen Safari.
************************************************************************************/
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>


////////////////////////////////////////////////////////////////////////////////
// Pure Functions
////////////////////////////////////////////////////////////////////////////////
/*@
requires 0 <= p;
ensures p < \result;
pure int ExpTwo(int p) = 0 < p ? 2 * ExpTwo(p - 1) : 1;
@*/

/*@
ensures |xs| == 0 ==> \result == 0;
ensures |xs| == 1 ==> \result == (xs.head);
pure int intsum(seq<int> xs) =
	0 < |xs| ? (xs.head) + intsum((xs.tail)) : 0;
@*/

/*@
requires n <= |xs|;
ensures n < 0 ==> |Take(xs, n)| == 0;
ensures 0 <= n ==> |Take(xs, n)| == n;
ensures (\forall int i; 0 <= i && i < n; xs[i] == get(Take(xs, n), i));
pure seq<int> Take(seq<int> xs, int n) =
	0 < n ? seq<int> { (xs.head) } + Take((xs.tail), n - 1) : seq<int> { };
@*/

/*@
requires 0 <= i && i <= |xs|;
ensures |\result| == |xs| - i;
ensures (\forall int j; 0 <= j && j < |\result|; \result[j] == intsum(Take(xs, i+j)));
pure seq<int> psum(seq<int> xs, int i) =
	i < |xs| ? seq<int> { intsum(Take(xs, i)) } + psum(xs, i + 1) : seq<int> { };
@*/

/*@
ensures |\result| == |xs|;
ensures (\forall int j; 0 <= j && j < |\result|; \result[j] == intsum(Take(xs, j)));
pure seq<int> psum2(seq<int> xs) = psum(xs, 0);
@*/

/*@
requires |xs| >= 0;
ensures |xs| == 0	==> \result == xs;
ensures |xs| == 1 ==> \result == xs;
ensures |xs| == 2 ==> \result == seq<int> { (xs.head) + xs.tail.head };
ensures |xs| % 2 == 0 ==> |\result| == |xs| / 2;
pure seq<int> implode(seq<int> xs) =
	1 < |xs| ? seq<int> { (xs.head) + xs.tail.head } + implode(xs.tail.tail) : xs;
@*/

/*@
requires 0 <= p;
pure int exp(int n, int p) = 0 < p ? n * exp(n, p - 1) : 1;
@*/

/*@
requires 0 <= n;
requires n < |xs|;
pure int get(seq<int> xs, int n) = xs[n];
@*/

/*@
requires k > 0;
requires |xs| == ExpTwo(k);
requires i >= 0 && i <= |xs|;
requires 1 <= lvl && lvl <= k;
requires stride == ExpTwo(lvl-1);
requires stride > 0 && stride < |xs|;
ensures |\result| == |xs| - i;
ensures (\forall int j; j >= 0 && j < |\result|; ((i < |xs|) && ((i+j) >= stride) && (((i+j) % (2*stride)) == (2*stride-1))) ==> \result[j] == xs[i+j] + xs[i+j - stride]);
ensures (\forall int j; j >= 0 && j < |\result|; ((i < |xs|) && (((i+j) < stride) || (((i+j) % (2*stride)) != (2*stride-1)))) ==> \result[j] == xs[i+j]);
pure seq<int> up(seq<int> xs, int stride, int i, int k, int lvl) =
	i < |xs| ? (
				((i % (2*stride)) == (2*stride-1) && (i >= stride)?
					seq<int> {xs[i] + xs[i-stride]} + up(xs, stride, i+1, k, lvl)
				:
					seq<int> {xs[i]} + up(xs, stride, i+1, k, lvl) ))
	:
		seq<int> {};
@*/

/*@
requires (\forall int i; 0 <= i && i < |xs|; xs[i] == 0 || xs[i] == 1);
ensures \result == intsum(xs);
ensures \result >= 0;
pure int count(seq<int> xs) =
	|xs| > 0 ? ( (xs.head) == 1 ? 1+count((xs.tail)) : count((xs.tail)) ) : 0;
@*/

/*@
requires |flags| == |input|;
requires (\forall int j; 0 <= j && j < |flags|; flags[j] == 0 || flags[j] == 1);
ensures |\result| == intsum(flags);
ensures 0 <= |\result| && |\result| <= |flags|;
pure seq<int> compact(seq<int> input, seq<int> flags) =
	0 < |input| ?
		(flags.head == 1 ? seq<int> { input.head } + compact(input.tail, flags.tail) : compact(input.tail, flags.tail))
	:
		seq<int> { };
@*/
////////////////////////////////////////////////////////////////////////////////////////Lemmas

/* The sum of a list of non-negative integers is itself non-negative. */
/*@
requires (\forall int i; 0 <= i && i < |xs|; 0 <= xs[i]);
ensures \result && 0 <= intsum(xs);
pure bool lemma_sum_nonneg(seq<int> xs);
@*/

/*@
ensures \result && intsum(seq<int> { }) == 0;
pure bool lemma_intsum_zero();
@*/

/*@
ensures \result && psum2(seq<int> { }) == seq<int> { };
pure bool lemma_psum_zero();
@*/

/*@
ensures \result && intsum(seq<int> { x }) == x;
pure bool lemma_intsum_single(int x);
@*/

/*@
requires |xs| == 1;
ensures \result && psum2(xs) == seq<int> {0};
pure bool lemma_psum_single(seq<int>  xs);
@*/


/*@
requires |xs| >= 0;
requires |ys| >= 0;
ensures \result && |xs| == 0 ==> intsum(xs + ys) == intsum(ys);
ensures \result && |ys| == 0 ==> intsum(xs + ys) == intsum(xs);
ensures \result && |xs + ys| == |xs| + |ys|;
ensures \result && intsum((xs.tail) + ys) == intsum((xs.tail)) + intsum(ys);
ensures \result && intsum(xs + ys) == intsum(xs) + intsum(ys);
pure bool lemma_intsum_app(seq<int> xs, seq<int> ys);
@*/

/*@
requires |xs| <= 1;
ensures \result && xs == implode(xs);
pure bool lemma_implode_base(seq<int> xs);
@*/


/*@
ensures \result && intsum(xs) == intsum(implode(xs));
pure bool lemma_implode_sum(seq<int> xs);
@*/

/*@
requires 0 < n;
ensures \result && ExpTwo(n) == 2 * ExpTwo(n - 1);
pure bool lemma_exp2_red_mult(int n);
@*/

/*@
requires 0 < n;
ensures \result && ExpTwo(n) / 2 == ExpTwo(n - 1);
pure bool lemma_exp2_red_div(int n);
@*/

/*@
requires 0 <= n;
ensures \result && 0 < ExpTwo(n);
pure bool lemma_exp2_positive(int n);
@*/

/*@
requires 0 <= i;
requires i <= j;
ensures \result && ExpTwo(i) <= ExpTwo(j);
pure bool lemma_exp2_leq(int i, int j);
@*/

/*@
requires i >= 0 && j >= 0;
requires ExpTwo(i) == ExpTwo(j);
ensures \result && i == j;
pure bool power_two_lemma(int i, int j);
@*/

/*@
requires |xs| % 2 == 0;
ensures \result && |implode(xs)| == |xs| / 2;
pure bool lemma_implode_length_mod_two(seq<int> xs);
@*/

/*@
requires 0 < n && |xs| == ExpTwo(n);
ensures \result && |implode(xs)| == ExpTwo(n - 1);
pure bool lemma_implode_red_exp2(seq<int> xs, int n);
@*/

/*@
requires 0 < i;
requires i < |xs|;
ensures \result && get((xs.tail), i - 1) == xs[i];
pure bool lemma_intseq_index_tail(seq<int> xs, int i);
@*/

/*@
requires |xs| % 2 == 0;
requires 0 <= i && i < |implode(xs)|;
requires (2 * i) < |xs|;
requires (2 * i + 1) < |xs|;
ensures \result && get(implode(xs), i) == xs[2 * i] + xs[2 * i + 1];
pure bool lemma_implode_get(seq<int> xs, int i);
@*/

/*@
requires |xs| % 2 == 0;
requires |implode(xs)| == |xs|/2;
ensures \result && (\forall int i; 0 <= i && i < |implode(xs)|; get(implode(xs), i) == xs[2 * i] + xs[2 * i + 1]);
pure bool lemma_implode_get_all(seq<int> xs);
@*/

/*@
requires |xs| == 2 * |ys|;
requires 0 <= |ys|;
requires (\forall int i; 0 <= i && i < |ys|; ys[i] == xs[2*i] + xs[2*i+1]);
ensures \result && ys == implode(xs);
pure bool lemma_implode_rel(seq<int> xs, seq<int> ys);
@*/

/*@
requires 0 <= i && i < |xs|;
ensures \result && get(psum2(xs), i) == intsum(Take(xs, i));
pure bool lemma_psum_get(seq<int> xs, int i);
@*/

/*@
ensures \result && (\forall int i; 0 <= i && i < |xs|; get(psum2(xs), i) == intsum(Take(xs, i)));
pure bool lemma_psum_get_all(seq<int> xs);
@*/

/*@
requires 0 < n && n <= |xs|;
ensures \result && Take(xs, n) == Take(xs, n - 1) + seq<int> { xs[n - 1] };
pure bool missing_lemma_2(seq<int> xs, int n);
@*/

/*@
requires |xs| % 2 == 0;
requires |ys| % 2 == 0;
ensures \result && implode(xs + ys) == implode(xs) + implode(ys);
pure bool missing_lemma_3(seq<int> xs, seq<int> ys);
@*/

/*@
ensures \result && xs + (ys + zs) == (xs + ys) + zs;
pure bool intseq_concat_assoc(seq<int> xs, seq<int> ys, seq<int> zs);
@*/

/*@
requires |xs| % 2 == 0;
requires 0 <= n && n < |implode(xs)|;
requires |implode(xs)| == |xs| / 2;
ensures \result && Take(implode(xs), n) == implode(Take(xs, 2 * n));
pure bool missing_lemma(seq<int> xs, int n);
@*/

/*@
requires |xs| % 2 == 0;
requires |implode(xs)| == |xs|/2;
requires 0 <= i && i < |implode(xs)|;
requires 2 * i < |xs|;
ensures \result && get(psum2(implode(xs)), i) == intsum(Take(xs, 2 * i));
pure bool lemma_psum_Take2(seq<int> xs, int i);
@*/

/*@
requires |xs| % 2 == 0;
requires |implode(xs)| == |xs|/2;
requires 0 <= i && i < |implode(xs)|;
requires 2 * i < |xs|;
ensures \result && get(psum2(implode(xs)), i) == get(psum2(xs), 2 * i);
pure bool lemma_get_psum_implode(seq<int> xs, int i);
@*/

/*@
requires 0 <= i;
requires 2 * i + 1 < |xs|;
ensures \result && get(psum2(xs), 2 * i + 1) == get(psum2(xs), 2 * i) + get(xs, 2 * i);
pure bool lemma_combine_psum(seq<int> xs, int i);
@*/

/*@
requires (\forall int j; 0 <= j && j < |xs|; xs[j] == 0 || xs[j] == 1);
ensures \result && intsum(xs) >= 0;
pure bool lemma_intsum_positive(seq<int> xs);
@*/

/*@
requires i >= 0;
requires i < |xs|;
requires (\forall int j; 0 <= j && j < i; xs[j] == 0 || xs[j] == 1);
ensures \result && i < |Take(xs, i)| ==> intsum(Take(xs, i)) >= 0;
pure bool lemma_intsum_flag(seq<int> xs, int i) = true;
@*/



/* assuming all elements in `xs` are non-negative, the sum of any sublist of `xs` will not be bigger than the sum of `xs`. */
/*@
requires n <= |xs|;
requires (\forall int i; 0 <= i && i < |xs|; 0 <= xs[i]);
ensures \result && 0 <= intsum(Take(xs, n)) && intsum(Take(xs, n)) <= intsum(xs);
pure bool lemma_take_sum(seq<int> xs, int n);
@*/

/*@
requires 0 <= n && n < |xs|;
requires (\forall int i; 0 <= i && i < |xs|; xs[i] == 0 || xs[i] == 1);
ensures \result && intsum(Take(xs, n)) < |xs|;
pure bool lemma_flags_take_size(seq<int> xs, int n);
@*/

/*@
requires 0 <= n && n < |xs|;
requires (\forall int i; 0 <= i && i < |xs|; xs[i] == 0 || xs[i] == 1);
requires xs[n] == 1;
ensures \result && intsum(Take(xs, n)) < intsum(xs);
pure bool lemma_sum_pos_flag(seq<int> xs, int n);
@*/

/*@
requires 0 <= n && n < |flags|;
requires flags[n] == 1;
requires |flags| == |input|;
requires (\forall int j; 0 <= j && j < |flags|; flags[j] == 0 || flags[j] == 1);
ensures \result && 0 <= intsum(Take(flags, n));
ensures \result && intsum(Take(flags, n)) < intsum(flags);
ensures \result && intsum(Take(flags, n)) < |flags|;
ensures \result && input[n] == get(compact(input, flags), intsum(Take(flags, n)));
ensures \result && input[n] == get(compact(input, flags), get(psum2(flags), n));
pure bool lemma_correctness(seq<int> input, seq<int> flags, int n);
@*/


////////////////////////////////////////////////////////////////////////////////
//Kernel
////////////////////////////////////////////////////////////////////////////////
/*@
context_everywhere flag_after_prefix != NULL;
context_everywhere flag_before_prefix != NULL;
context_everywhere input != NULL;
context_everywhere output != NULL;
context_everywhere k == 10;
context_everywhere M == 8;
context_everywhere blockDim.x == ExpTwo(k);
context_everywhere gridDim.x == 1;
requires \ltid < (ExpTwo(k)+2-1)/2 ==> \pointer_index(flag_after_prefix, 2*\ltid, write);
requires \ltid < (ExpTwo(k)-1+2-1)/2 ==> \pointer_index(flag_after_prefix, 2*\ltid+1, write);
requires (\ltid >= 0 && \ltid <= 0) ==> (\forall* int i; 0 <= i && i < ExpTwo(k) && (i + 1) % 1 != 0; \pointer_index(flag_after_prefix, i, write));
requires \pointer_index(input, \ltid, 1\2);
requires \pointer_index(flag_before_prefix, \ltid, 1\2);
requires flag_before_prefix[\ltid] == 0 || flag_before_prefix[\ltid] == 1;
requires \ltid < ExpTwo(M) ==> \pointer_index(output, \ltid, write);
@*/
__global__ void CUDA_Kernel_Stream_Compaction(int* input, int* output, int* flag_before_prefix, int* flag_after_prefix, int k, int M)
{
  int tid = threadIdx.x;
  //@ assert tid == \ltid;

  //@ ghost seq<int> flag_seq;
  //@ assume |flag_seq| == ExpTwo(k);
  //@ assume (\forall int i; 0 <= i && i < ExpTwo(k); flag_seq[i] == 0 || flag_seq[i] == 1);
  //@ assume (flag_seq[tid] == 0 || flag_seq[tid] == 1) && count(flag_seq) == ExpTwo(M);
	//@ assume (2 * tid  < ExpTwo(k)) ==> flag_after_prefix[2 * tid] == flag_seq[2 * tid];
	//@ assume (2 * tid + 1 < ExpTwo(k)) ==> flag_after_prefix[2 * tid + 1] == flag_seq[2 * tid + 1];

  //@ assume flag_seq[tid] == flag_before_prefix[tid];

  //@ assume  2 * tid  < ExpTwo(k) ==> flag_after_prefix[2 * tid] == 0 || flag_after_prefix[2 * tid] == 1;
	//@ assume 2 * tid + 1 < ExpTwo(k) ==> flag_after_prefix[2 * tid + 1] == 0 || flag_after_prefix[2 * tid + 1] == 1;

  //@ ghost seq<int> inp;
  //@ assume |inp| == ExpTwo(k) && inp[tid] == input[tid];


	int indicator = 2 * tid + 1;
	int stride = 1;

	int lvl = 1;

	//@ ghost seq<seq<int> > Matrix_UP = seq<seq<int> > { flag_seq };
	//@ assert (\forall int i; 0 < i && i < lvl; Matrix_UP[i] == up(Matrix_UP[i - 1], stride/ExpTwo(lvl-i), 0, k, i));
	//@ ghost seq<seq<int> > Matrix = seq<seq<int> > { flag_seq };

  /*@
	loop_invariant k > 0;
	loop_invariant tid >= 0 && tid < ExpTwo(k);
	loop_invariant stride > 0;
	loop_invariant 1 <= lvl;
	loop_invariant stride == ExpTwo(lvl-1);
	loop_invariant lvl <= k+1;
	loop_invariant indicator + 1 == ExpTwo(lvl)*(tid+1);
	loop_invariant indicator + 1 == 2*stride*(tid+1);
	loop_invariant indicator > 0;
	loop_invariant stride <= ExpTwo(k);
	loop_invariant indicator < ExpTwo(k) ==> \pointer_index(flag_after_prefix, indicator, 1);
	loop_invariant indicator < ExpTwo(k) && indicator >= stride ==> \pointer_index(flag_after_prefix, indicator - stride, 1);
	loop_invariant tid==0 ==> (\forall* int i; 0 <= i && i < ExpTwo(k) && (i + 1) % stride != 0; \pointer_index(flag_after_prefix, i, 1));
	loop_invariant (tid==0 && (stride == ExpTwo(k))) ==> \pointer_index(flag_after_prefix, ExpTwo(k) - 1, 1);
	loop_invariant |Matrix_UP| == lvl;
	loop_invariant (\forall int i; 0 <= i && i < lvl; |Matrix_UP[i]| == ExpTwo(k));
	loop_invariant lvl == 1 ==> Matrix_UP[lvl - 1] == flag_seq;
	loop_invariant lvl > 1 && lvl < |Matrix_UP| ==> Matrix_UP[lvl] == up(Matrix_UP[lvl - 1], (stride/2) - 1, 0, k, lvl - 1);
	loop_invariant indicator < ExpTwo(k) ==> Matrix_UP[lvl - 1][indicator] == flag_after_prefix[indicator];
	loop_invariant indicator < ExpTwo(k) && indicator >= stride ==> Matrix_UP[lvl - 1][indicator - stride] == flag_after_prefix[indicator - stride];
	loop_invariant lvl == k+1 ==> Matrix_UP[lvl-1][ExpTwo(k) - 1] == intsum(flag_seq);
	loop_invariant lvl == k+1 ==> Matrix_UP[lvl-1][(ExpTwo(k) - 1)/2] == intsum(Take(flag_seq, |flag_seq|/2));
	loop_invariant |Matrix| == lvl;
	loop_invariant (\forall int i; 0 <= i && i < lvl; 0 <= |Matrix[i]| && |Matrix[i]| <= ExpTwo(k));
	loop_invariant (\forall int i; 0 <= i && i < lvl; |Matrix[i]| == ExpTwo(k - i));
	loop_invariant (\forall int i; 0 < i && i < lvl; Matrix[i] == implode(Matrix[i - 1]));
	loop_invariant (\forall int i; 0 <= i && i < lvl; intsum(Matrix[i]) == intsum(flag_seq));
	loop_invariant Matrix[0] == flag_seq;
	loop_invariant indicator < ExpTwo(k) && 2 * tid + 1 < |Matrix[lvl - 1]| ==> flag_after_prefix[indicator] == Matrix[lvl - 1][2 * tid + 1];
	loop_invariant indicator < ExpTwo(k) && indicator >= stride && 2 * tid < |Matrix[lvl - 1]| ==> flag_after_prefix[indicator - stride] == Matrix[lvl - 1][2 * tid];
	@*/
  while(stride < ExpTwo(k))
	{


		if(indicator < ExpTwo(k) && indicator >= stride)
		{
			//@ assert 2 * tid + 1 < |Matrix[lvl - 1]| ==> flag_after_prefix[indicator] == Matrix[lvl - 1][2 * tid + 1];
			//@ assert 2 * tid < |Matrix[lvl - 1]| ==> flag_after_prefix[indicator - stride] == Matrix[lvl - 1][2 * tid];
			flag_after_prefix[indicator] = flag_after_prefix[indicator] + flag_after_prefix[indicator - stride];
			//@ assert 2 * tid + 1 < |Matrix[lvl - 1]| ==> flag_after_prefix[indicator] == Matrix[lvl - 1][2 * tid + 1] + Matrix[lvl - 1][2 * tid];
		}

		//@ assert lemma_implode_length_mod_two(Matrix[lvl - 1]);
		//@ assert lemma_implode_sum(Matrix[lvl - 1]);
		//@ assert lemma_implode_get_all(Matrix[lvl - 1]);


		//@ ghost Matrix = Matrix + seq<seq<int> > { implode(Matrix[lvl - 1]) };

		//@ ghost tid < |implode(Matrix[lvl - 1])| ? (lemma_implode_get(Matrix[lvl - 1], tid) && (2 * tid + 1 < |Matrix[lvl - 1]| ==> get(implode(Matrix[lvl - 1]), tid) == Matrix[lvl - 1][2 * tid] + Matrix[lvl - 1][2 * tid + 1]) && (indicator < ExpTwo(k) && indicator >= stride ==> flag_after_prefix[indicator] == Matrix[lvl - 1][2 * tid + 1] + Matrix[lvl - 1][2 * tid]) && (Matrix[lvl] == implode(Matrix[lvl - 1])) && (indicator < ExpTwo(k) && indicator >= stride ==> flag_after_prefix[indicator] == Matrix[lvl][tid])) : true;




    /*if(tid < |implode(Matrix[lvl - 1])|){
		lemma_implode_get(Matrix[lvl - 1], tid);
		assert 2 * tid + 1 < |Matrix[lvl - 1]| ==> get(implode(Matrix[lvl - 1]), tid) == Matrix[lvl - 1][2 * tid] + Matrix[lvl - 1][2 * tid + 1];
		assert indicator < flag_after_prefix.length && indicator >= stride ==> flag_after_prefix[indicator] == Matrix[lvl - 1][2 * tid + 1] + Matrix[lvl - 1][2 * tid];
		assert Matrix[lvl] == implode(Matrix[lvl - 1]);
		assert indicator < flag_after_prefix.length && indicator >= stride ==> flag_after_prefix[indicator] == Matrix[lvl][tid];
		}*/


    /*@
			context k > 0;
			context 1 <= lvl && lvl <= k;
			context |Matrix| == lvl + 1;
			requires tid >= 0 && tid < ExpTwo(k);
			requires stride == ExpTwo(lvl-1);
			requires stride > 0 && stride < ExpTwo(k);
			requires indicator + 1 == ExpTwo(lvl)*(tid+1);
			requires indicator + 1 == 2*stride*(tid+1);
			requires indicator > 0;
			requires indicator < ExpTwo(k) ==> \pointer_index(flag_after_prefix, indicator, 1);
			requires indicator < ExpTwo(k) && indicator >= stride ==> \pointer_index(flag_after_prefix, indicator - stride, 1);
			requires tid==0 ==> (\forall* int i; 0 <= i && i < ExpTwo(k) && (i + 1) % stride != 0; \pointer_index(flag_after_prefix, i, 1));
			ensures tid >= 0 && tid < ExpTwo(k);
			ensures 2 * stride == ExpTwo(lvl);
			ensures 2 * stride > 0 && 2 * stride <= ExpTwo(k);
			ensures 2 * indicator + 2 == ExpTwo(lvl+1)*(tid+1);
			ensures 2 * indicator + 2 == 2*stride*(tid+1);
			ensures 2 * indicator + 1 > 0;
			ensures 2 * indicator + 1 < ExpTwo(lvl) ==> \pointer_index(flag_after_prefix, 2 * indicator + 1, 1);
			ensures 2 * indicator + 1 < ExpTwo(lvl) && 2 * indicator + 1 >= 2 * stride  ==> \pointer_index(flag_after_prefix, 2 * indicator + 1 - 2 * stride, 1);
			ensures tid==0 ==> (\forall* int i; 0 <= i && i < ExpTwo(lvl) && (i + 1) % (2 * stride) != 0; \pointer_index(flag_after_prefix, i, 1));
			ensures (tid==0 && (2 * stride == ExpTwo(lvl))) ==> \pointer_index(flag_after_prefix, ExpTwo(k) - 1, 1);
    @*/
    __syncthreads();

		//@ ghost Matrix_UP = Matrix_UP + seq<seq<int> > { up(Matrix_UP[lvl - 1], stride, 0, k, lvl) };
		//@ assert (indicator < ExpTwo(k)) && (indicator >= stride) ==> Matrix_UP[lvl][indicator] == Matrix_UP[lvl - 1][indicator] + Matrix_UP[lvl - 1][indicator-stride];
		indicator = 2 * indicator + 1;
		stride = 2 * stride;
		lvl = lvl + 1;
		//@ assert (\forall int i; 0 < i && i < lvl; Matrix_UP[i] == up(Matrix_UP[i - 1], stride/ExpTwo(lvl-i), 0, k, i));

		//@ assert stride == ExpTwo(lvl-1);
		//@ assert lemma_exp2_red_mult(lvl);
		//@ assert ExpTwo(lvl) == 2 * ExpTwo(lvl - 1);
		//@ assert 2*stride == ExpTwo(lvl);
		//@ assert indicator + 1 == ExpTwo(lvl)*(tid+1);
		//@ assert indicator + 1 == 2*stride*(tid+1);


	}

	//@ assert stride == ExpTwo(lvl-1);
	//@ assert ExpTwo(lvl-1) == ExpTwo(k);
	//@ assert stride == ExpTwo(k);
	//@ assert power_two_lemma(lvl-1, k);
	//@ assert lvl == k + 1;
	//@ assert indicator < ExpTwo(k) ==> Matrix_UP[lvl - 1][indicator] == flag_after_prefix[indicator];
	//@ assert |Matrix| == lvl;
	//@ assert (\forall int i; 0 <= i && i < k + 1; |Matrix[i]| == ExpTwo(k - i));
	//@ assert (\forall int i; 0 < i && i < k + 1; Matrix[i] == implode(Matrix[i - 1]));
	//@ assert (\forall int i; 0 <= i && i < k + 1; intsum(Matrix[i]) == intsum(flag_seq));
	//@ assert |Matrix[k]| == 1;
	//@ assert lemma_intsum_single(Matrix[k][0]);
	//@ assert intsum(Matrix[k]) == intsum(flag_seq);
	//@ assert Matrix[k] == seq<int>{intsum(flag_seq)};
	//@ assert Matrix[0] == flag_seq;
	//@ assert (\forall int i; 0 <= i && i < k + 1; 0 < |Matrix[i]| && |Matrix[i]| <= ExpTwo(k));

/////////////////////////////////////////////////////////////////////////////////

  //@ assert indicator < ExpTwo(k) && indicator >= stride && 2 * tid < |Matrix[lvl - 1]| ==> flag_after_prefix[indicator - stride] == Matrix[lvl - 1][2 * tid];

  /*@
		context k > 0;
		context |Matrix_UP| == k + 1;
        context |Matrix| == k + 1;
		context lvl == k + 1;
		context stride == ExpTwo(k);
		context indicator + 1 == ExpTwo(lvl)*(tid+1);
		context indicator + 1 == 2*stride*(tid+1);
		context indicator > 0;
		context stride > 0 ;
		requires indicator < ExpTwo(k) ==> \pointer_index(flag_after_prefix, indicator, 1);
		requires indicator < ExpTwo(k) && indicator >= stride  ==> \pointer_index(flag_after_prefix, indicator - stride, 1);
		requires tid==0 ==> (\forall* int i; 0 <= i && i < ExpTwo(k) && (i + 1) % stride != 0; \pointer_index(flag_after_prefix, i, 1));
		requires (tid==0 && (stride == ExpTwo(k))) ==> \pointer_index(flag_after_prefix, ExpTwo(k) - 1, 1);
		requires (\forall int i; 0 <= i && i <= k; |Matrix_UP[i]| == ExpTwo(k));
    requires (\forall int i; 0 <= i && i < lvl; |Matrix[i]| == ExpTwo(k - i));
    requires (\forall int i; 0 <= i && i < lvl; 0 <= |Matrix[i]| && |Matrix[i]| <= ExpTwo(k));
		requires indicator < ExpTwo(k) && indicator >= stride ==> Matrix_UP[lvl - 1][indicator] == flag_after_prefix[indicator];
		requires indicator < ExpTwo(k) && indicator >= stride ==> Matrix_UP[lvl - 1][indicator - stride] == flag_after_prefix[indicator - stride];
    requires indicator < ExpTwo(k) && indicator >= stride && 2 * tid < |Matrix[lvl - 1]| ==> flag_after_prefix[indicator - stride] == Matrix[lvl - 1][2 * tid];
		context tid >= 0 && tid < ExpTwo(k);
		//ensures stride == ExpTwo(k) / 2;
		//ensures indicator == ExpTwo(k) * tid + ExpTwo(k) - 1;
		//ensures stride > 0 ;
		//ensures indicator > 0;
		ensures ExpTwo(k) * \ltid + ExpTwo(k) - 1 < ExpTwo(k) ==> \pointer_index(flag_after_prefix, ExpTwo(k) * \ltid + ExpTwo(k) - 1, 1);
		ensures ExpTwo(k) * \ltid + ExpTwo(k) - 1 < ExpTwo(k) && ExpTwo(k) * \ltid + ExpTwo(k) - 1 >= ExpTwo(k) / 2  ==> \pointer_index(flag_after_prefix, ExpTwo(k) * \ltid + ExpTwo(k) - 1 - ExpTwo(k) / 2, 1);
		ensures tid==0 ==> (\forall* int i; 0 <= i && i < ExpTwo(k) && (i + 1) % (ExpTwo(k) / 2) != 0; \pointer_index(flag_after_prefix, i, 1));
    ensures (\forall int i; 0 <= i && i <= k; |Matrix_UP[i]| == ExpTwo(k));
    ensures (\forall int i; 0 <= i && i < lvl; |Matrix[i]| == ExpTwo(k - i));
    ensures (\forall int i; 0 <= i && i < lvl; 0 <= |Matrix[i]| && |Matrix[i]| <= ExpTwo(k));
		//ensures ExpTwo(k) * \ltid + ExpTwo(k) - 1 < ExpTwo(k) ==> Matrix_UP[lvl - 1][ExpTwo(k) * \ltid + ExpTwo(k) - 1] == flag_after_prefix[ExpTwo(k) * \ltid + ExpTwo(k) - 1];
		//ensures ExpTwo(k) * \ltid + ExpTwo(k) - 1 < ExpTwo(k) && ExpTwo(k) * \ltid + ExpTwo(k) - 1 >= ExpTwo(k) / 2 ==> Matrix_UP[lvl - 1][ExpTwo(k) * \ltid + ExpTwo(k) - 1 - ExpTwo(k) / 2] == flag_after_prefix[ExpTwo(k) * \ltid + ExpTwo(k) - 1 - ExpTwo(k) / 2];
    //ensures 2 * tid < |Matrix[lvl-2]| && ExpTwo(k) * \ltid + ExpTwo(k) - 1 < ExpTwo(k) && ExpTwo(k) * \ltid + ExpTwo(k) - 1 >= ExpTwo(k) / 2 ==> flag_after_prefix[ExpTwo(k) * \ltid + ExpTwo(k) - 1 - ExpTwo(k) / 2] == get(Matrix[lvl-2], 2 * tid);
  @*/
  __syncthreads();

  // (unstability) These come from the las three postconditions in the previous barrier:
		//@ assume ExpTwo(k) * tid + ExpTwo(k) - 1 < ExpTwo(k) ==> Matrix_UP[lvl - 1][ExpTwo(k) * tid + ExpTwo(k) - 1] == flag_after_prefix[ExpTwo(k) * tid + ExpTwo(k) - 1];
		//@ assume ExpTwo(k) * tid + ExpTwo(k) - 1 < ExpTwo(k) && ExpTwo(k) * tid + ExpTwo(k) - 1 >= ExpTwo(k) / 2 ==> Matrix_UP[lvl - 1][ExpTwo(k) * tid + ExpTwo(k) - 1 - ExpTwo(k) / 2] == flag_after_prefix[ExpTwo(k) * tid + ExpTwo(k) - 1 - ExpTwo(k) / 2];
    //@ assume 2 * tid < |Matrix[lvl-2]| && ExpTwo(k) * tid + ExpTwo(k) - 1 < ExpTwo(k) && ExpTwo(k) * tid + ExpTwo(k) - 1 >= ExpTwo(k) / 2 ==> flag_after_prefix[ExpTwo(k) * tid + ExpTwo(k) - 1 - ExpTwo(k) / 2] == get(Matrix[lvl-2], 2 * tid);

///////////////////////////////////////////////////////////////////////////////////////	Down

	indicator = ExpTwo(k) * tid + ExpTwo(k) - 1; // flag_after_prefix.length * tid + flag_after_prefix.length - 1;
	stride = ExpTwo(k) / 2; // flag_after_prefix.length / 2;
	lvl = k - 1; //lvl - 2;
	int temp;
	//@ ghost seq<int> temp_seq = seq<int> { 0 };

	//@ assert ExpTwo(k) * tid + ExpTwo(k) - 1 < ExpTwo(k) ==> Matrix_UP[lvl + 1][indicator] == flag_after_prefix[indicator];
	//@ assert ExpTwo(k) * tid + ExpTwo(k) - 1 < ExpTwo(k) && ExpTwo(k) * tid + ExpTwo(k) - 1 >= ExpTwo(k) / 2 ==> Matrix_UP[lvl + 1][indicator - stride] == flag_after_prefix[indicator - stride];



	if(indicator < ExpTwo(k))
	{
		flag_after_prefix[indicator] = 0;
	}





  /*@
  loop_invariant k > 0;
  loop_invariant tid >= 0 && tid < ExpTwo(k);
  loop_invariant lvl <= k - 1;
  loop_invariant lvl >= -1;
  loop_invariant lvl >= 0 ==> stride == ExpTwo(lvl);
  loop_invariant lvl == -1 ==> stride == 0;
  loop_invariant stride == 0 ==> lvl == -1;
  loop_invariant stride >= 0;
  loop_invariant indicator >= 0;
  loop_invariant indicator+1 == ExpTwo(lvl+1)*(tid+1);
  loop_invariant indicator < ExpTwo(k) ==> \pointer_index(flag_after_prefix, indicator, 1);
  loop_invariant lvl >= 0 && indicator < ExpTwo(k) && indicator >= stride ==> \pointer_index(flag_after_prefix, indicator - stride, 1);
  loop_invariant (tid==0 && stride > 0) ==> (\forall* int i; 0 <= i && i < ExpTwo(k) && (i + 1) % stride != 0; \pointer_index(flag_after_prefix, i, 1));
  //loop_invariant lvl == -1 ==> \pointer_index(flag_after_prefix, tid, 1);
  //loop_invariant lvl == -1 ==> indicator == tid;
  //loop_invariant indicator == tid ==> lvl == -1;
  loop_invariant |temp_seq| == ExpTwo(k - (lvl + 1));
  loop_invariant 0 < |temp_seq| && |temp_seq| <= ExpTwo(k);
  loop_invariant temp_seq == psum2(Matrix[lvl + 1]);
  loop_invariant (\forall int i; 0 <= i && i < k + 1; 0 < |Matrix[i]| && |Matrix[i]| <= ExpTwo(k));
  loop_invariant (\forall int i; 0 <= i && i < k + 1; |Matrix[i]| == ExpTwo(k - i));
  loop_invariant (\forall int i; 0 <= i && i < k + 1; intsum(Matrix[i]) == intsum(flag_seq));
  loop_invariant (\forall int i; 0 < i && i < k + 1; Matrix[i] == implode(Matrix[i - 1]));
  loop_invariant Matrix[0] == flag_seq;
  loop_invariant Matrix[k] == seq<int>{ intsum(flag_seq) };
  loop_invariant tid < |temp_seq| && indicator < ExpTwo(k) ==> temp_seq[tid] == flag_after_prefix[indicator];
  loop_invariant lvl >= 0 && 2 * tid < |Matrix[lvl]| && indicator < ExpTwo(k) && indicator >= stride ==> flag_after_prefix[indicator - stride] == get(Matrix[lvl], 2 * tid);
  @*/
  while(stride >= 1)
  {
  	if(indicator < ExpTwo(k) && indicator >= stride)
  	{

  		//@ assert tid < |temp_seq| ==> temp_seq[tid] == flag_after_prefix[indicator];
  		temp = flag_after_prefix[indicator];
  		//@ assert tid < |temp_seq| ==> temp == temp_seq[tid];
  		flag_after_prefix[indicator] = flag_after_prefix[indicator] + flag_after_prefix[indicator - stride];
  		//@ assert tid < |temp_seq| ==> flag_after_prefix[indicator] == temp_seq[tid] + flag_after_prefix[indicator - stride];

  		//@ assert 2 * tid < |Matrix[lvl]| ==> flag_after_prefix[indicator - stride] == get(Matrix[lvl], 2 * tid);
  		//@ assert 2 * tid < |Matrix[lvl]| && tid < |temp_seq| ==> flag_after_prefix[indicator] == temp_seq[tid] + get(Matrix[lvl], 2 * tid);
  		//@ assert tid < |Matrix[lvl + 1]| && tid < |temp_seq| ==> temp_seq[tid] == get(psum2(Matrix[lvl + 1]), tid);
  		//@ assert tid < |Matrix[lvl + 1]| && 2 * tid < |Matrix[lvl]| ==> flag_after_prefix[indicator] == get(psum2(Matrix[lvl + 1]), tid) + get(Matrix[lvl], 2 * tid);
  		//@ assert Matrix[lvl + 1] == implode(Matrix[lvl]);
  		//@ assert tid < |implode(Matrix[lvl])| && 2 * tid < |Matrix[lvl]| ==> flag_after_prefix[indicator] == get(psum2(implode(Matrix[lvl])), tid) + get(Matrix[lvl], 2 * tid);
      //@ ghost tid < |implode(Matrix[lvl])| ? lemma_get_psum_implode(Matrix[lvl], tid) : true;
  		/*if(tid < |implode(Matrix[lvl])|){
  			lemma_get_psum_implode(Matrix[lvl], tid);
  		}*/
  		//@ assert tid < |implode(Matrix[lvl])| && 2 * tid < |Matrix[lvl]| ==> get(psum2(implode(Matrix[lvl])), tid) == get(psum2(Matrix[lvl]), 2 * tid);
  		//@ assert 2 * tid < |Matrix[lvl]| ==> flag_after_prefix[indicator] == get(psum2(Matrix[lvl]), 2 * tid) + get(Matrix[lvl], 2 * tid);
  		//@ ghost 2 * tid + 1 < |Matrix[lvl]| ? lemma_combine_psum(Matrix[lvl], tid) : true;
      /*if(2 * tid + 1 < |Matrix[lvl]|){
  		  lemma_combine_psum(Matrix[lvl], tid);
  		}*/
  		//@ assert 2 * tid + 1 < |Matrix[lvl]| ==> get(psum2(Matrix[lvl]), 2 * tid + 1) == get(psum2(Matrix[lvl]), 2 * tid) + get(Matrix[lvl], 2 * tid);
  		//@ assert 2 * tid + 1 < |Matrix[lvl]| ==> flag_after_prefix[indicator] == get(psum2(Matrix[lvl]), 2 * tid + 1);

  		//@ assert tid < |temp_seq| ==> temp == temp_seq[tid];
  		flag_after_prefix[indicator - stride] = temp;
  		//@ assert tid < |temp_seq| ==> flag_after_prefix[indicator - stride] == temp_seq[tid];

  		//@ assert tid < |Matrix[lvl + 1]| && tid < |temp_seq| ==> temp_seq[tid] == get(psum2(Matrix[lvl + 1]), tid);
  		//@ assert Matrix[lvl + 1] == implode(Matrix[lvl]);
  		//@ assert tid < |implode(Matrix[lvl])| && tid < |temp_seq| ==> temp_seq[tid] == get(psum2(implode(Matrix[lvl])), tid);
  		//@ ghost tid < |implode(Matrix[lvl])| ? lemma_get_psum_implode(Matrix[lvl], tid) : true;
      /*if(tid < |implode(Matrix[lvl])|){
  			lemma_get_psum_implode(Matrix[lvl], tid);
  		}*/

  		//@ assert tid < |implode(Matrix[lvl])| && 2 * tid < |Matrix[lvl]| ==> get(psum2(implode(Matrix[lvl])), tid) == get(psum2(Matrix[lvl]), 2 * tid);
  		//@ assert 2 * tid < |Matrix[lvl]| && tid < |temp_seq| ==> temp_seq[tid] == get(psum2(Matrix[lvl]), 2 * tid);
  		//@ assert 2 * tid < |Matrix[lvl]| ==> flag_after_prefix[indicator - stride] == get(psum2(Matrix[lvl]), 2 * tid);

  	}

  	//@ ghost temp_seq = psum2(Matrix[lvl]);

  	//@ assert 2 * tid < |temp_seq| && indicator < ExpTwo(k) && indicator >= stride ==> flag_after_prefix[indicator - stride] == temp_seq[2 * tid];
  	//@ assert 2 * tid + 1 < |temp_seq| && indicator < ExpTwo(k) && indicator >= stride ==> flag_after_prefix[indicator] == temp_seq[2 * tid + 1];


    /*@
  		context lvl >= 0 && lvl <= k - 1;
  		requires tid >= 0 && tid < ExpTwo(k);
        context |temp_seq| == ExpTwo(k - lvl);
        context 0 < |temp_seq| && |temp_seq| <= ExpTwo(k);
        context |Matrix| == k + 1;
      //context lvl - 1 == -1 ==> (indicator - 1) / 2 == \ltid;
      //context (indicator - 1) / 2 == \ltid ==> lvl - 1 == -1;
  		requires indicator >= 0;
  		requires stride >= 1 ;
  		requires stride == ExpTwo(lvl);
  		requires indicator+1 == ExpTwo(lvl+1)*(\ltid+1);
  		requires indicator < ExpTwo(k) ==> \pointer_index(flag_after_prefix, indicator, 1);
  		requires indicator < ExpTwo(k) && indicator >= stride  ==> \pointer_index(flag_after_prefix, indicator - stride, 1);
  		requires tid==0 ==> (\forall* int i; 0 <= i && i < ExpTwo(k) && (i + 1) % stride != 0; \pointer_index(flag_after_prefix, i, 1));
      //requires 2 * tid < |temp_seq| && indicator < ExpTwo(k) && indicator >= stride ==> flag_after_prefix[indicator - stride] == temp_seq[2 * tid];
      //requires 2 * tid + 1 < |temp_seq| && indicator < ExpTwo(k) && indicator >= stride ==> flag_after_prefix[indicator] == temp_seq[2 * tid + 1];
      requires (\forall int i; 0 <= i && i < k + 1; |Matrix[i]| == ExpTwo(k - i));
      requires (\forall int i; 0 <= i && i < k + 1; 0 <= |Matrix[i]| && |Matrix[i]| <= ExpTwo(k));
      //requires 2 * tid < |Matrix[lvl]| && indicator < ExpTwo(k) && indicator >= stride ==> flag_after_prefix[indicator - stride] == get(psum2(Matrix[lvl]), 2 * tid);
  		ensures tid >= 0 && tid < ExpTwo(k);
  		ensures lvl-1 >= 0 ==> stride / 2 == ExpTwo(lvl - 1);
  		ensures lvl-1 == -1 ==> stride / 2 == 0;
      ensures stride / 2 == 0  ==> lvl-1 == -1;
  		ensures stride / 2 >= 0;
  		ensures (indicator - 1) / 2 >= 0;
  		ensures (indicator - 1) / 2+1 == ExpTwo(lvl)*(tid+1);
  		ensures (indicator - 1) / 2 < ExpTwo(k) ==> \pointer_index(flag_after_prefix, (indicator - 1) / 2, 1);
  		ensures lvl-1 >= 0 && (indicator - 1) / 2 < ExpTwo(k) && (indicator - 1) / 2 >= stride / 2  ==> \pointer_index(flag_after_prefix, (indicator - 1) / 2 - stride / 2, 1);
  		ensures (tid==0 && stride/2 > 0) ==> (\forall* int i; 0 <= i && i < ExpTwo(k) && (i + 1) % (stride/2) != 0; \pointer_index(flag_after_prefix, i, 1));
      //ensures tid < |temp_seq| && (indicator - 1) / 2 < ExpTwo(k) ==> temp_seq[tid] == flag_after_prefix[(indicator - 1) / 2];
      ensures (\forall int i; 0 <= i && i < k + 1; |Matrix[i]| == ExpTwo(k - i));
      ensures (\forall int i; 0 <= i && i < k + 1; 0 <= |Matrix[i]| && |Matrix[i]| <= ExpTwo(k));
      //ensures lvl-1 >= 0 && 2 * tid < |Matrix[lvl-1]| && (indicator - 1) / 2 < ExpTwo(k) && (indicator - 1) / 2 >= stride / 2 ==> flag_after_prefix[(indicator - 1) / 2 - stride / 2] == get(Matrix[lvl-1], 2 * tid);
    @*/
    __syncthreads();

  	//@ assume tid < |temp_seq| && (indicator - 1) / 2 < ExpTwo(k) ==> temp_seq[tid] == flag_after_prefix[(indicator - 1) / 2];
  	//@ assume lvl-1 >= 0 && 2 * tid < |Matrix[lvl-1]| && (indicator - 1) / 2 < ExpTwo(k) && (indicator - 1) / 2 >= stride / 2 ==> flag_after_prefix[(indicator - 1) / 2 - stride / 2] == get(Matrix[lvl-1], 2 * tid);

  	indicator = (indicator - 1) / 2;
  	stride = stride / 2;
  	lvl = lvl - 1;


  }


	//@ assert indicator == tid;
  //@ assert tid >= 0 && tid < ExpTwo(k);
	//@ assert temp_seq == psum2(Matrix[0]);
	//@ assert Matrix[0] == flag_seq;
	//@ assert temp_seq == psum2(flag_seq);
  //@ assert |temp_seq| == ExpTwo(k);
	//@ assert temp_seq[tid] == flag_after_prefix[indicator];
	//@ assert flag_after_prefix[indicator] == get(psum2(flag_seq), tid);
	//@ assert lemma_intsum_flag(flag_seq, indicator);
	//@ assert count(flag_seq) == ExpTwo(M);
  //@ assert intsum(flag_seq) == ExpTwo(M);
  //@ assert flag_seq[tid] == 1 ? lemma_sum_pos_flag(flag_seq, tid) : true;
  //@ assert lemma_take_sum(flag_seq, tid);
  //@ assert flag_seq[tid] == flag_before_prefix[indicator];
	//@ assert flag_after_prefix[indicator] >= 0 && flag_after_prefix[indicator] <= ExpTwo(M);
	//@ assert flag_before_prefix[indicator] == 1 ==> flag_after_prefix[indicator] >= 0 && flag_after_prefix[indicator] < ExpTwo(M);


  /*@
    requires indicator == tid;
    requires tid >= 0 && tid < ExpTwo(k);
    requires tid < ExpTwo(M) ==> \pointer_index(output, tid, 1);
    requires \pointer_index(input, tid, 1\2);
    requires \pointer_index(flag_before_prefix, tid, 1\2);
    requires \pointer_index(flag_after_prefix, tid, 1);
    requires |temp_seq| == ExpTwo(k);
    requires temp_seq == psum2(flag_seq);
    //requires temp_seq[tid] == flag_after_prefix[indicator];
    requires |flag_seq| == ExpTwo(k);
    requires flag_seq[tid] == flag_before_prefix[indicator];
    requires |inp| == ExpTwo(k);
    requires (inp[tid] == input[tid]);
    ensures indicator == tid;
    ensures tid >= 0 && tid < ExpTwo(k);
    ensures |temp_seq| == ExpTwo(k);
    ensures temp_seq == psum2(flag_seq);
    ensures \pointer_index(input, tid, 1\2);
    ensures \pointer_index(flag_before_prefix, tid, 1\2);
    ensures \pointer_index(flag_after_prefix, tid, 1\2);
  	ensures flag_before_prefix[tid] == 1 ==> flag_after_prefix[tid] >= 0 && flag_after_prefix[tid] < ExpTwo(M);
  	ensures flag_before_prefix[tid] == 1 ==> \pointer_index(output, flag_after_prefix[tid], 1);
    //ensures temp_seq[tid] == flag_after_prefix[indicator];
    ensures |flag_seq| == ExpTwo(k);
    ensures flag_seq[tid] == flag_before_prefix[indicator];
    ensures |inp| == ExpTwo(k);
    ensures (inp[tid] == input[tid]);
  @*/
  __syncthreads();


	if(flag_before_prefix[tid] == 1){

		output[flag_after_prefix[tid]] = input[tid];
    //@ assert (output[flag_after_prefix[tid]] == input[tid]);

	}

   //@ assert flag_before_prefix[tid] == 1 ==> (output[flag_after_prefix[tid]] == input[tid]);

	//@ ghost seq<int> temporary;

	//@ ghost temporary = compact(inp, flag_seq);
  //@ assert temporary == compact(inp, flag_seq);
	//@ assert |temporary| == ExpTwo(M);

  //@ assert intsum(Take(flag_seq, tid)) >= 0;
  //@ assume temp_seq[tid] == flag_after_prefix[indicator];
  //@ assert flag_after_prefix[tid] >= 0;

  //@ assert \pointer_index(input, tid, 1\2);
  //@ assert \pointer_index(flag_before_prefix, tid, 1\2);
  //@ assert \pointer_index(flag_after_prefix, tid, 1\2);
  //@ assert flag_before_prefix[tid] == 1 ==> \pointer_index(output, flag_after_prefix[tid], 1);

  //@ assert flag_before_prefix[tid] == flag_seq[tid];

  //@ assert flag_before_prefix[tid] == 1 ==> (lemma_correctness(inp, flag_seq, tid));

  //@ assert flag_before_prefix[tid] == 1 ==> inp[tid] == get(compact(inp, flag_seq), intsum(Take(flag_seq, tid)));

  // assert temporary == compact(inp, flag_seq);

  //@ assert flag_before_prefix[tid] == 1 ==> (inp[tid] == get(temporary, intsum(Take(flag_seq, tid))));

  //@ assert (inp[tid] == input[tid]);

  //@ assert flag_before_prefix[tid] == 1 ==> (input[tid] == get(temporary, flag_after_prefix[tid]));

  // assert flag_before_prefix[tid] == 1 ==> (output[flag_after_prefix[tid]] == input[tid]);

  //@ assert flag_before_prefix[tid] == 1 ==> (output[flag_after_prefix[tid]] == get(temporary, flag_after_prefix[tid]));


}