#include "hip/hip_runtime.h"
//:: case GPGPUExample
/***********************************************************************************
Created by Mohsen Safari.
************************************************************************************/
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

///////// ///////////////////////////////////////////////////////////////////////
//Kernel
////////////////////////////////////////////////////////////////////////////////
/*@
context_everywhere N == blockDim.x;
context \pointer_index(g_array1, \gtid, write);
context \pointer_index(g_array2, \gtid, write);
ensures g_array1[\gtid] <= \old(g_array1[\gtid]) && g_array1[\gtid] <= g_array2[\gtid];
ensures g_array1[\gtid] == \old(g_array1[\gtid]) || g_array1[\gtid] == g_array2[\gtid];
@*/
__global__ void CUDAKernel(int* g_array1, int* g_array2, int N)
{
  int tid = blockIdx.x * N + threadIdx.x;
  //@ assert tid == \gtid;
  atomicMin(g_array1 + tid, g_array2[tid]) /*@ then { assert false && false; } */;
}

//@ ensures \pointer(\result, N, write);
int *vercorsMallocInt(int N);
void vercorsFreeInt(int *ar);
//@ ensures \pointer(\result, N, write);
int *vercorsCudaMallocInt(int N);
void vercorsCudaFreeInt(int *addr);
void vercorsCudaMemcpyInt(int *tgt, int *src, int N, int direction);

////////////////////////////////////////////////////////////////////////////////
// Main Program
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{

  int N = 1024; // size of the array

  // allocate host memory
  // int* host_array1 = (int*) malloc(sizeof(int)*N);
  // int* host_array2 = (int*) malloc(sizeof(int)*N);
  int* host_array1 = vercorsMallocInt(N);
  int* host_array2 = vercorsMallocInt(N);
    
  // initalize the memory
  //@ loop_invariant 0 <= i && i <= N;
  //@ loop_invariant \pointer(host_array1, N, write);
  //@ loop_invariant \pointer(host_array2, N, write);
  //@ loop_invariant (\forall int j; 0 <= j && j < i; host_array1[j] == j && host_array2[j] == N-j);
  for(unsigned int i = 0; i < N; i++) 
  {
    host_array1[i] = i;
    host_array2[i] = N-i;
  }   


  //Copy the arrays to device memory
  int* device_array1;
  device_array1 = vercorsCudaMallocInt(N);
  vercorsCudaMemcpyInt( device_array1, host_array1, N, hipMemcpyHostToDevice) ;

  int* device_array2;
  device_array2 = vercorsCudaMallocInt(N);
  vercorsCudaMemcpyInt( device_array2, host_array2, N, hipMemcpyHostToDevice) ;

    
  //setup execution parameters
  int num_of_blocks = 1;
  int num_of_threads_per_block = N;
  
  //dim3  grid( num_of_blocks, 1, 1); //grid has three parameters to indicate the dimensions. Here we have one dimensional grid (of blocks). It can be one, two or three dimensions.
  //dim3  threads( num_of_threads_per_block, 1, 1); //threads indicates the dimensions of one block. Here each block has one dimension (of threads). It can be one, two or three dimensions.

  //start the timer
  hipEvent_t begin, end;
  int time;
  begin = hipEventCreate();
  end = hipEventCreate();
  hipEventRecord(begin, 0);

  //Kernel launch
  CUDAKernel<<< /*grid*/num_of_blocks, /*threads*/num_of_threads_per_block/*, 0*/ >>>(device_array1, device_array2, N);

  //Stop the Timer
  hipEventRecord(end, 0);
  hipEventSynchronize(end);
  time = hipEventElapsedTime(begin, end);
  hipEventDestroy(begin);
  hipEventDestroy(end);
  
  // copy result from device to host
  vercorsCudaMemcpyInt(host_array1, device_array1, N, hipMemcpyDeviceToHost);

  //print kernel execution time 
  // printf( "Processing time: %d (ms)\n", time);

  // cleanup memory
  vercorsFreeInt(host_array1);
  vercorsFreeInt(host_array2);
  vercorsCudaFreeInt(device_array1);
  vercorsCudaFreeInt(device_array2);

  return 0;
       
}