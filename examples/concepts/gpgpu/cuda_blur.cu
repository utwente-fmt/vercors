//:: cases SharedCuda
//:: tool silicon
//:: verdict Pass

#include <hip/hip_runtime.h>

/*@
  context blockDim.x == 32 && blockDim.y == 1 && blockDim.z == 1;
  context gridDim.x > 0 && gridDim.y == 1 && gridDim.z == 1;
  context in != NULL && out != NULL;
  context \pointer_length(in) == n;
  context \pointer_length(out) == n-2;
  context blockDim.x * gridDim.x >= n;
  context (\forall* int i; 0<=i && i<n; Perm(&in[i], write \ (blockDim.x * gridDim.x)));

  context \gtid<n-2 ==> Perm(&out[\gtid], write);

  context \shared_mem_size(s) == blockDim.x+2;
  requires Perm(&s[\ltid], write);
  requires threadIdx.x < 2 ==> Perm(&s[\ltid + blockDim.x], write);

  ensures \gtid<n-2 ==>out[\gtid] == (in[\gtid] + in[\gtid+1] + in[\gtid+2])/3;
  
@*/
__global__ void blur_x(int* in, int* out, int n) {
  extern __shared__ int s[];
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if(tid < n) {
    s[threadIdx.x] = in[tid];
  }
  if(threadIdx.x < 2 && tid+blockDim.x < n){
    s[blockDim.x+threadIdx.x] = in[tid+blockDim.x];
  }

  /*@
    context (\forall* int i; 0<=i && i<n; Perm(&in[i], write \ (blockDim.x * gridDim.x)));
    context blockIdx.x * blockDim.x + threadIdx.x<n-2 ==> Perm(&out[blockIdx.x * blockDim.x + threadIdx.x], write);

    requires Perm(&s[threadIdx.x], write);
    requires threadIdx.x < 2 ==> Perm(&s[threadIdx.x + blockDim.x], write);

    requires blockIdx.x * blockDim.x + threadIdx.x < n
      ==> s[threadIdx.x] == in[blockIdx.x * blockDim.x + threadIdx.x];
    requires threadIdx.x < 2 && blockIdx.x * blockDim.x + threadIdx.x+blockDim.x < n
      ==> s[threadIdx.x+blockDim.x] == in[blockIdx.x * blockDim.x + threadIdx.x+blockDim.x];


    ensures (\forall* int i; 0 <= i && i < blockDim.x+2; Perm(&s[i], write \ (blockDim.x+2)));

    ensures 
      (\forall int i; 0 <= i && i < blockDim.x+2 && blockIdx.x*blockDim.x + i <n
        ==> s[i] == in[blockIdx.x * blockDim.x+i]);
  @*/
  __syncthreads();

  if(tid < n-2) {
    out[tid] = (s[threadIdx.x] + s[threadIdx.x+1] + s[threadIdx.x+2])/3;
  }
}