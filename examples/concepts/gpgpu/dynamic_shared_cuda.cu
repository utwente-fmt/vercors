//:: cases DynamicSharedCuda
//:: tool silicon
//:: verdict Pass

#include <hip/hip_runtime.h>

/*@
  context blockDim.x == 32 && blockDim.y == 1 && blockDim.z == 1;
  context gridDim.x > 0 && gridDim.y == 1 && gridDim.z == 1;

  context in != NULL && out != NULL;
  context \pointer_length(in) == 1;
  context \pointer_length(out) == n;
  context n > 0;
  context blockDim.x * gridDim.x >= n;
  context Perm(&in[0], write \ (blockDim.x * gridDim.x));
  context \gtid<n ==> Perm(&out[\gtid], write);

  context \shared_mem_size(s) == 1;
  requires \ltid == 0 ==> Perm(&s[0], write);

  ensures \gtid<n ==> out[\gtid] == \old(out[\gtid]) + in[0];
@*/
__global__ void blur_x(int* in, int* out, int n) {
  extern __shared__ int s[];
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if(threadIdx.x == 0) {
    s[threadIdx.x] = in[0];
  }

  /*@
    context Perm(&in[0], write \ (blockDim.x * gridDim.x));
    context blockIdx.x * blockDim.x + threadIdx.x<n ==> Perm(&out[blockIdx.x * blockDim.x + threadIdx.x], write);
    context blockIdx.x * blockDim.x + threadIdx.x<n ==> \old(out[blockIdx.x * blockDim.x + threadIdx.x]) == out[blockIdx.x * blockDim.x + threadIdx.x];

    requires threadIdx.x == 0 ==> Perm(&s[0], write);
    requires threadIdx.x == 0 ==> s[0] == in[0];

    ensures Perm(&s[0], write \ blockDim.x);

    ensures s[0] == in[0];
  @*/
  __syncthreads();

  if(tid < n) {
    out[tid] += s[0];
  }
}